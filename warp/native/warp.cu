#include "hip/hip_runtime.h"
/** Copyright (c) 2022 NVIDIA CORPORATION.  All rights reserved.
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include "warp.h"
#include "scan.h"
#include "cuda_util.h"
#include "error.h"

#include <hip/hiprtc.h>
#include <nvPTXCompiler.h>

#include <algorithm>
#include <iterator>
#include <list>
#include <map>
#include <string>
#include <unordered_map>
#include <unordered_set>
#include <vector>

#define check_nvrtc(code) (check_nvrtc_result(code, __FILE__, __LINE__))
#define check_nvptx(code) (check_nvptx_result(code, __FILE__, __LINE__))

bool check_nvrtc_result(hiprtcResult result, const char* file, int line)
{
    if (result == HIPRTC_SUCCESS)
        return true;

    const char* error_string = hiprtcGetErrorString(result);
    fprintf(stderr, "Warp NVRTC compilation error %u: %s (%s:%d)\n", unsigned(result), error_string, file, line);
    return false;
}

bool check_nvptx_result(nvPTXCompileResult result, const char* file, int line)
{
    if (result == NVPTXCOMPILE_SUCCESS)
        return true;

    const char* error_string;
    switch (result)
    {
    case NVPTXCOMPILE_ERROR_INVALID_COMPILER_HANDLE:
        error_string = "Invalid compiler handle";
        break;
    case NVPTXCOMPILE_ERROR_INVALID_INPUT:
        error_string = "Invalid input";
        break;
    case NVPTXCOMPILE_ERROR_COMPILATION_FAILURE:
        error_string = "Compilation failure";
        break;
    case NVPTXCOMPILE_ERROR_INTERNAL:
        error_string = "Internal error";
        break;
    case NVPTXCOMPILE_ERROR_OUT_OF_MEMORY:
        error_string = "Out of memory";
        break;
    case NVPTXCOMPILE_ERROR_COMPILER_INVOCATION_INCOMPLETE:
        error_string = "Incomplete compiler invocation";
        break;
    case NVPTXCOMPILE_ERROR_UNSUPPORTED_PTX_VERSION:
        error_string = "Unsupported PTX version";
        break;
    default:
        error_string = "Unknown error";
        break;
    }

    fprintf(stderr, "Warp PTX compilation error %u: %s (%s:%d)\n", unsigned(result), error_string, file, line);
    return false;
}


struct DeviceInfo
{
    static constexpr int kNameLen = 128;

    hipDevice_t device = -1;
    hipUUID uuid = {0};
    int ordinal = -1;
    int pci_domain_id = -1;
    int pci_bus_id = -1;
    int pci_device_id = -1;
    char name[kNameLen] = "";
    int arch = 0;
    int is_uva = 0;
    int is_mempool_supported = 0;
    hipCtx_t primary_context = NULL;
};

struct ContextInfo
{
    DeviceInfo* device_info = NULL;

    // the current stream, managed from Python (see cuda_context_set_stream() and cuda_context_get_stream())
    hipStream_t stream = NULL;
};

struct CaptureInfo
{
    hipStream_t stream = NULL;  // the main stream where capture begins and ends
    uint64_t id = 0;  // unique capture id from CUDA
    bool external = false;  // whether this is an external capture
};

struct StreamInfo
{
    hipEvent_t cached_event = NULL;  // event used for stream synchronization (cached to avoid creating temporary events)
    CaptureInfo* capture = NULL;  // capture info (only if started on this stream)
};

struct GraphInfo
{
    std::vector<void*> unfreed_allocs;
};

// Information for graph allocations that are not freed by the graph.
// These allocations have a shared ownership:
// - The graph instance allocates/maps the memory on each launch, even if the user reference is released.
// - The user reference must remain valid even if the graph is destroyed.
// The memory will be freed once the user reference is released and the graph is destroyed.
struct GraphAllocInfo
{
    uint64_t capture_id = 0;
    void* context = NULL;
    bool ref_exists = false;  // whether user reference still exists
    bool graph_destroyed = false;  // whether graph instance was destroyed
};

// Information used when deferring deallocations.
struct FreeInfo
{
    void* context = NULL;
    void* ptr = NULL;
    bool is_async = false;
};

static std::unordered_map<hipFunction_t, std::string> g_kernel_names;

// cached info for all devices, indexed by ordinal
static std::vector<DeviceInfo> g_devices;

// maps hipDevice_t to DeviceInfo
static std::map<hipDevice_t, DeviceInfo*> g_device_map;

// cached info for all known contexts
static std::map<hipCtx_t, ContextInfo> g_contexts;

// cached info for all known streams (including registered external streams)
static std::unordered_map<hipStream_t, StreamInfo> g_streams;

// Ongoing graph captures registered using wp.capture_begin().
// This maps the capture id to the stream where capture was started.
// See cuda_graph_begin_capture(), cuda_graph_end_capture(), and free_device_async().
static std::unordered_map<uint64_t, CaptureInfo*> g_captures;

// Memory allocated during graph capture requires special handling.
// See alloc_device_async() and free_device_async().
static std::unordered_map<void*, GraphAllocInfo> g_graph_allocs;

// Memory that cannot be freed immediately gets queued here.
// Call free_deferred_allocs() to release.
static std::vector<FreeInfo> g_deferred_free_list;


void cuda_set_context_restore_policy(bool always_restore)
{
    ContextGuard::always_restore = always_restore;
}

int cuda_get_context_restore_policy()
{
    return int(ContextGuard::always_restore);
}

int cuda_init()
{
    if (!init_cuda_driver())
        return -1;

    int device_count = 0;
    if (check_cu(cuDeviceGetCount_f(&device_count)))
    {
        g_devices.resize(device_count);

        for (int i = 0; i < device_count; i++)
        {
            hipDevice_t device;
            if (check_cu(cuDeviceGet_f(&device, i)))
            {
                // query device info
                g_devices[i].device = device;
                g_devices[i].ordinal = i;
                check_cu(cuDeviceGetName_f(g_devices[i].name, DeviceInfo::kNameLen, device));
                check_cu(cuDeviceGetUuid_f(&g_devices[i].uuid, device));
                check_cu(cuDeviceGetAttribute_f(&g_devices[i].pci_domain_id, hipDeviceAttributePciDomainId, device));
                check_cu(cuDeviceGetAttribute_f(&g_devices[i].pci_bus_id, hipDeviceAttributePciBusId, device));
                check_cu(cuDeviceGetAttribute_f(&g_devices[i].pci_device_id, hipDeviceAttributePciDeviceId, device));
                check_cu(cuDeviceGetAttribute_f(&g_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                check_cu(cuDeviceGetAttribute_f(&g_devices[i].is_mempool_supported, hipDeviceAttributeMemoryPoolsSupported, device));
                int major = 0;
                int minor = 0;
                check_cu(cuDeviceGetAttribute_f(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(cuDeviceGetAttribute_f(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                g_devices[i].arch = 10 * major + minor;

                g_device_map[device] = &g_devices[i];
            }
            else
            {
                return -1;
            }
        }
    }
    else
    {
        return -1;
    }

    // initialize default timing state
    static CudaTimingState default_timing_state(0, NULL);
    g_cuda_timing_state = &default_timing_state;

    return 0;
}


static inline hipCtx_t get_current_context()
{
    hipCtx_t ctx;
    if (check_cu(cuCtxGetCurrent_f(&ctx)))
        return ctx;
    else
        return NULL;
}

static inline hipStream_t get_current_stream(void* context=NULL)
{
    return static_cast<hipStream_t>(cuda_context_get_stream(context));
}

static ContextInfo* get_context_info(hipCtx_t ctx)
{
    if (!ctx)
    {
        ctx = get_current_context();
        if (!ctx)
            return NULL;
    }

    auto it = g_contexts.find(ctx);
    if (it != g_contexts.end())
    {
        return &it->second;
    }
    else
    {
        // previously unseen context, add the info
        ContextGuard guard(ctx, true);

        hipDevice_t device;
        if (check_cu(cuCtxGetDevice_f(&device)))
        {
            DeviceInfo* device_info = g_device_map[device];

            // workaround for https://nvbugspro.nvidia.com/bug/4456003
            if (device_info->is_mempool_supported)
            {
                void* dummy = NULL;
                check_cuda(hipMallocAsync(&dummy, 1, NULL));
                check_cuda(hipFreeAsync(dummy, NULL));
            }

            ContextInfo context_info;
            context_info.device_info = device_info;
            auto result = g_contexts.insert(std::make_pair(ctx, context_info));
            return &result.first->second;
        }
    }

    return NULL;
}

static inline ContextInfo* get_context_info(void* context)
{
    return get_context_info(static_cast<hipCtx_t>(context));
}

static inline StreamInfo* get_stream_info(hipStream_t stream)
{
    auto it = g_streams.find(stream);
    if (it != g_streams.end())
        return &it->second;
    else
        return NULL;
}

static void deferred_free(void* ptr, void* context, bool is_async)
{
    FreeInfo free_info;
    free_info.ptr = ptr;
    free_info.context = context ? context : get_current_context();
    free_info.is_async = is_async;
    g_deferred_free_list.push_back(free_info);
}

static int free_deferred_allocs(void* context = NULL)
{
    if (g_deferred_free_list.empty() || !g_captures.empty())
        return 0;

    int num_freed_allocs = 0;
    for (auto it = g_deferred_free_list.begin(); it != g_deferred_free_list.end(); /*noop*/)
    {
        const FreeInfo& free_info = *it;

        // free the pointer if it matches the given context or if the context is unspecified
        if (free_info.context == context || !context)
        {
            ContextGuard guard(free_info.context);

            if (free_info.is_async)
            {
                // this could be a regular stream-ordered allocation or a graph allocation
                hipError_t res = hipFreeAsync(free_info.ptr, NULL);
                if (res != hipSuccess)
                {
                    if (res == hipErrorInvalidValue)
                    {
                        // This can happen if we try to release the pointer but the graph was
                        // never launched, so the memory isn't mapped.
                        // This is fine, so clear the error.
                        hipGetLastError();
                    }
                    else
                    {
                        // something else went wrong, report error
                        check_cuda(res);
                    }
                }
            }
            else
            {
                check_cuda(hipFree(free_info.ptr));
            }

            ++num_freed_allocs;

            it = g_deferred_free_list.erase(it);
        }
        else
        {
            ++it;
        }
    }

    return num_freed_allocs;
}

static void CUDART_CB on_graph_destroy(void* user_data)
{
    if (!user_data)
        return;

    GraphInfo* graph_info = static_cast<GraphInfo*>(user_data);

    for (void* ptr : graph_info->unfreed_allocs)
    {
        auto alloc_iter = g_graph_allocs.find(ptr);
        if (alloc_iter != g_graph_allocs.end())
        {
            GraphAllocInfo& alloc_info = alloc_iter->second;
            if (alloc_info.ref_exists)
            {
                // unreference from graph so the pointer will be deallocated when the user reference goes away
                alloc_info.graph_destroyed = true;
            }
            else
            {
                // the pointer can be freed, but we can't call CUDA functions in this callback, so defer it
                deferred_free(ptr, alloc_info.context, true);
                g_graph_allocs.erase(alloc_iter);
            }
        }
    }

    delete graph_info;
}

static inline const char* get_cuda_kernel_name(void* kernel)
{
    hipFunction_t cuda_func = static_cast<hipFunction_t>(kernel);
    auto name_iter = g_kernel_names.find((hipFunction_t)cuda_func);
    if (name_iter != g_kernel_names.end())
        return name_iter->second.c_str();
    else
        return "unknown_kernel";
}


void* alloc_pinned(size_t s)
{
    void* ptr = NULL;
    check_cuda(hipHostMalloc(&ptr, s));
    return ptr;
}

void free_pinned(void* ptr)
{
    hipHostFree(ptr);
}

void* alloc_device(void* context, size_t s)
{
    int ordinal = cuda_context_get_device_ordinal(context);

    // use stream-ordered allocator if available
    if (cuda_device_is_mempool_supported(ordinal))
        return alloc_device_async(context, s);
    else
        return alloc_device_default(context, s);
}

void free_device(void* context, void* ptr)
{
    int ordinal = cuda_context_get_device_ordinal(context);

    // use stream-ordered allocator if available
    if (cuda_device_is_mempool_supported(ordinal))
        free_device_async(context, ptr);
    else
        free_device_default(context, ptr);
}

void* alloc_device_default(void* context, size_t s)
{
    ContextGuard guard(context);

    void* ptr = NULL;
    check_cuda(hipMalloc(&ptr, s));

    return ptr;
}

void free_device_default(void* context, void* ptr)
{
    ContextGuard guard(context);

    // check if a capture is in progress
    if (g_captures.empty())
    {
        check_cuda(hipFree(ptr));
    }
    else
    {
        // we must defer the operation until graph captures complete
        deferred_free(ptr, context, false);
    }
}

void* alloc_device_async(void* context, size_t s)
{
    // stream-ordered allocations don't rely on the current context,
    // but we set the context here for consistent behaviour
    ContextGuard guard(context);

    ContextInfo* context_info = get_context_info(context);
    if (!context_info)
        return NULL;

    hipStream_t stream = context_info->stream;

    void* ptr = NULL;
    check_cuda(hipMallocAsync(&ptr, s, stream));

    if (ptr)
    {
        // if the stream is capturing, the allocation requires special handling
        if (cuda_stream_is_capturing(stream))
        {
            // check if this is a known capture
            uint64_t capture_id = get_capture_id(stream);
            auto capture_iter = g_captures.find(capture_id);
            if (capture_iter != g_captures.end())
            {
                // remember graph allocation details
                GraphAllocInfo alloc_info;
                alloc_info.capture_id = capture_id;
                alloc_info.context = context ? context : get_current_context();
                alloc_info.ref_exists = true;  // user reference created and returned here
                alloc_info.graph_destroyed = false;  // graph not destroyed yet
                g_graph_allocs[ptr] = alloc_info;
            }
        }
    }

    return ptr;
}

void free_device_async(void* context, void* ptr)
{
    // stream-ordered allocators generally don't rely on the current context,
    // but we set the context here for consistent behaviour
    ContextGuard guard(context);

    // NB: Stream-ordered deallocations are tricky, because the memory could still be used on another stream
    // or even multiple streams.  To avoid use-after-free errors, we need to ensure that all preceding work
    // completes before releasing the memory.  The strategy is different for regular stream-ordered allocations
    // and allocations made during graph capture.  See below for details.

    // check if this allocation was made during graph capture
    auto alloc_iter = g_graph_allocs.find(ptr);
    if (alloc_iter == g_graph_allocs.end())
    {
        // Not a graph allocation.
        // Check if graph capture is ongoing.
        if (g_captures.empty())
        {
            // hipFreeAsync on the null stream does not block or trigger synchronization, but it postpones
            // the deallocation until a synchronization point is reached, so preceding work on this pointer
            // should safely complete.
            check_cuda(hipFreeAsync(ptr, NULL));
        }
        else
        {
            // We must defer the free operation until graph capture completes.
            deferred_free(ptr, context, true);
        }
    }
    else
    {
        // get the graph allocation details
        GraphAllocInfo& alloc_info = alloc_iter->second;

        uint64_t capture_id = alloc_info.capture_id;

        // check if the capture is still active
        auto capture_iter = g_captures.find(capture_id);
        if (capture_iter != g_captures.end())
        {
            // Add a mem free node.  Use all current leaf nodes as dependencies to ensure that all prior
            // work completes before deallocating.  This works with both Warp-initiated and external captures
            // and avoids the need to explicitly track all streams used during the capture.
            CaptureInfo* capture = capture_iter->second;
            hipGraph_t graph = get_capture_graph(capture->stream);
            std::vector<hipGraphNode_t> leaf_nodes;
            if (graph && get_graph_leaf_nodes(graph, leaf_nodes))
            {
                hipGraphNode_t free_node;
                check_cuda(hipGraphAddMemFreeNode(&free_node, graph, leaf_nodes.data(), leaf_nodes.size(), ptr));
            }

            // we're done with this allocation, it's owned by the graph
            g_graph_allocs.erase(alloc_iter);
        }
        else
        {
            // the capture has ended
            // if the owning graph was already destroyed, we can free the pointer now
            if (alloc_info.graph_destroyed)
            {
                if (g_captures.empty())
                {
                    // try to free the pointer now
                    hipError_t res = hipFreeAsync(ptr, NULL);
                    if (res == hipErrorInvalidValue)
                    {
                        // This can happen if we try to release the pointer but the graph was
                        // never launched, so the memory isn't mapped.
                        // This is fine, so clear the error.
                        hipGetLastError();
                    }
                    else
                    {
                        // check for other errors
                        check_cuda(res);
                    }
                }
                else
                {
                    // We must defer the operation until graph capture completes.
                    deferred_free(ptr, context, true);
                }

                // we're done with this allocation
                g_graph_allocs.erase(alloc_iter);
            }
            else
            {
                // graph still exists
                // unreference the pointer so it will be deallocated once the graph instance is destroyed
                alloc_info.ref_exists = false;
            }
        }
    }
}

bool memcpy_h2d(void* context, void* dest, void* src, size_t n, void* stream)
{
    ContextGuard guard(context);

    hipStream_t hip_stream;
    if (stream != WP_CURRENT_STREAM)
        hip_stream = static_cast<hipStream_t>(stream);
    else
        hip_stream = get_current_stream(context);

    begin_cuda_range(WP_TIMING_MEMCPY, hip_stream, context, "memcpy HtoD");

    bool result = check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyHostToDevice, hip_stream));

    end_cuda_range(WP_TIMING_MEMCPY, hip_stream);

    return result;
}

bool memcpy_d2h(void* context, void* dest, void* src, size_t n, void* stream)
{
    ContextGuard guard(context);

    hipStream_t hip_stream;
    if (stream != WP_CURRENT_STREAM)
        hip_stream = static_cast<hipStream_t>(stream);
    else
        hip_stream = get_current_stream(context);

    begin_cuda_range(WP_TIMING_MEMCPY, hip_stream, context, "memcpy DtoH");

    bool result = check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToHost, hip_stream));

    end_cuda_range(WP_TIMING_MEMCPY, hip_stream);

    return result;
}

bool memcpy_d2d(void* context, void* dest, void* src, size_t n, void* stream)
{
    ContextGuard guard(context);

    hipStream_t hip_stream;
    if (stream != WP_CURRENT_STREAM)
        hip_stream = static_cast<hipStream_t>(stream);
    else
        hip_stream = get_current_stream(context);

    begin_cuda_range(WP_TIMING_MEMCPY, hip_stream, context, "memcpy DtoD");

    bool result = check_cuda(hipMemcpyAsync(dest, src, n, hipMemcpyDeviceToDevice, hip_stream));

    end_cuda_range(WP_TIMING_MEMCPY, hip_stream);

    return result;
}

bool memcpy_p2p(void* dst_context, void* dst, void* src_context, void* src, size_t n, void* stream)
{
    // ContextGuard guard(context);

    hipStream_t hip_stream;
    if (stream != WP_CURRENT_STREAM)
        hip_stream = static_cast<hipStream_t>(stream);
    else
        hip_stream = get_current_stream(dst_context);

    // Notes:
    // - cuMemcpyPeerAsync() works fine with both regular and pooled allocations (hipMalloc() and hipMallocAsync(), respectively)
    //   when not capturing a graph.
    // - cuMemcpyPeerAsync() is not supported during graph capture, so we must use hipMemcpyAsync() with kind=hipMemcpyDefault.
    // - hipMemcpyAsync() works fine with regular allocations, but doesn't work with pooled allocations
    //   unless mempool access has been enabled.
    // - There is no reliable way to check if mempool access is enabled during graph capture,
    //   because hipMemPoolGetAccess() cannot be called during graph capture.
    // - CUDA will report error 1 (invalid argument) if hipMemcpyAsync() is called but mempool access is not enabled.

    if (!cuda_stream_is_capturing(stream))
    {
        begin_cuda_range(WP_TIMING_MEMCPY, hip_stream, get_stream_context(stream), "memcpy PtoP");

        bool result = check_cu(cuMemcpyPeerAsync_f(
            (hipDeviceptr_t)dst, (hipCtx_t)dst_context,
            (hipDeviceptr_t)src, (hipCtx_t)src_context,
            n, hip_stream));

        end_cuda_range(WP_TIMING_MEMCPY, hip_stream);

        return result;
    }
    else
    {
        hipError_t result = hipSuccess;

        // hipMemcpyAsync() is sensitive to the bound context to resolve pointer locations.
        // If fails with hipErrorInvalidValue if it cannot resolve an argument.
        // We first try the copy in the destination context, then if it fails we retry in the source context.
        // The hipErrorInvalidValue error doesn't cause graph capture to fail, so it's ok to retry.
        // Since this trial-and-error shenanigans only happens during capture, there
        // is no perf impact when the graph is launched.
        // For bonus points, this approach simplifies memory pool access requirements.
        // Access only needs to be enabled one way, either from the source device to the destination device
        // or vice versa.  Sometimes, when it's really quiet, you can actually hear my genius.
        {
            // try doing the copy in the destination context
            ContextGuard guard(dst_context);
            result = hipMemcpyAsync(dst, src, n, hipMemcpyDefault, hip_stream);

            if (result != hipSuccess)
            {
                // clear error in destination context
                hipGetLastError();

                // try doing the copy in the source context
                ContextGuard guard(src_context);
                result = hipMemcpyAsync(dst, src, n, hipMemcpyDefault, hip_stream);

                // clear error in source context
                hipGetLastError();
            }
        }

        // If the copy failed, try to detect if mempool allocations are involved to generate a helpful error message.
        if (!check_cuda(result))
        {
            if (result == hipErrorInvalidValue && src != NULL && dst != NULL)
            {
                // check if either of the pointers was allocated from a mempool
                void* src_mempool = NULL;
                void* dst_mempool = NULL;
                cuPointerGetAttribute_f(&src_mempool, HIP_POINTER_ATTRIBUTE_MEMPOOL_HANDLE, (hipDeviceptr_t)src);
                cuPointerGetAttribute_f(&dst_mempool, HIP_POINTER_ATTRIBUTE_MEMPOOL_HANDLE, (hipDeviceptr_t)dst);
                hipGetLastError();  // clear any errors
                // check if either of the pointers was allocated during graph capture
                auto src_alloc = g_graph_allocs.find(src);
                auto dst_alloc = g_graph_allocs.find(dst);
                if (src_mempool != NULL || src_alloc != g_graph_allocs.end() ||
                    dst_mempool != NULL || dst_alloc != g_graph_allocs.end())
                {
                    wp::append_error_string("*** CUDA mempool allocations were used in a peer-to-peer copy during graph capture.");
                    wp::append_error_string("*** This operation fails if mempool access is not enabled between the peer devices.");
                    wp::append_error_string("*** Either enable mempool access between the devices or use the default CUDA allocator");
                    wp::append_error_string("*** to pre-allocate the arrays before graph capture begins.");
                }
            }

            return false;
        }

        return true;
    }
}


__global__ void memset_kernel(int* dest, int value, size_t n)
{
    const size_t tid = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
    
    if (tid < n)
    {
        dest[tid] = value;
    }
}

void memset_device(void* context, void* dest, int value, size_t n)
{
    ContextGuard guard(context);

    if (true)// ((n%4) > 0)
    {
        hipStream_t stream = get_current_stream();

        begin_cuda_range(WP_TIMING_MEMSET, stream, context, "memset");

        // for unaligned lengths fallback to CUDA memset
        check_cuda(hipMemsetAsync(dest, value, n, stream));

        end_cuda_range(WP_TIMING_MEMSET, stream);
    }
    else
    {
        // custom kernel to support 4-byte values (and slightly lower host overhead)
        const size_t num_words = n/4;
        wp_launch_device(WP_CURRENT_CONTEXT, memset_kernel, num_words, ((int*)dest, value, num_words));
    }
}

// fill memory buffer with a value: generic memtile kernel using memcpy for each element
__global__ void memtile_kernel(void* dst, const void* src, size_t srcsize, size_t n)
{
    size_t tid = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
    if (tid < n)
    {
        memcpy((int8_t*)dst + srcsize * tid, src, srcsize);
    }
}

// this should be faster than memtile_kernel, but requires proper alignment of dst
template <typename T>
__global__ void memtile_value_kernel(T* dst, T value, size_t n)
{
    size_t tid = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) + static_cast<size_t>(threadIdx.x);
    if (tid < n)
    {
        dst[tid] = value;
    }
}

void memtile_device(void* context, void* dst, const void* src, size_t srcsize, size_t n)
{
    ContextGuard guard(context);

    size_t dst_addr = reinterpret_cast<size_t>(dst);
    size_t src_addr = reinterpret_cast<size_t>(src);

    // try memtile_value first because it should be faster, but we need to ensure proper alignment
    if (srcsize == 8 && (dst_addr & 7) == 0 && (src_addr & 7) == 0)
    {
        int64_t* p = reinterpret_cast<int64_t*>(dst);
        int64_t value = *reinterpret_cast<const int64_t*>(src);
        wp_launch_device(WP_CURRENT_CONTEXT, memtile_value_kernel, n, (p, value, n));
    }
    else if (srcsize == 4 && (dst_addr & 3) == 0 && (src_addr & 3) == 0)
    {
        int32_t* p = reinterpret_cast<int32_t*>(dst);
        int32_t value = *reinterpret_cast<const int32_t*>(src);
        wp_launch_device(WP_CURRENT_CONTEXT, memtile_value_kernel, n, (p, value, n));
    }
    else if (srcsize == 2 && (dst_addr & 1) == 0 && (src_addr & 1) == 0)
    {
        int16_t* p = reinterpret_cast<int16_t*>(dst);
        int16_t value = *reinterpret_cast<const int16_t*>(src);
        wp_launch_device(WP_CURRENT_CONTEXT, memtile_value_kernel, n, (p, value, n));
    }
    else if (srcsize == 1)
    {
        check_cuda(hipMemset(dst, *reinterpret_cast<const int8_t*>(src), n));
    }
    else
    {
        // generic version

        // copy value to device memory
        // TODO: use a persistent stream-local staging buffer to avoid allocs?
        void* src_devptr = alloc_device(WP_CURRENT_CONTEXT, srcsize);
        check_cuda(hipMemcpyAsync(src_devptr, src, srcsize, hipMemcpyHostToDevice, get_current_stream()));

        wp_launch_device(WP_CURRENT_CONTEXT, memtile_kernel, n, (dst, src_devptr, srcsize, n));

        free_device(WP_CURRENT_CONTEXT, src_devptr);

    }
}


static __global__ void array_copy_1d_kernel(void* dst, const void* src,
                                        int dst_stride, int src_stride,
                                        const int* dst_indices, const int* src_indices,
                                        int n, int elem_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        int src_idx = src_indices ? src_indices[i] : i;
        int dst_idx = dst_indices ? dst_indices[i] : i;
        const char* p = (const char*)src + src_idx * src_stride;
        char* q = (char*)dst + dst_idx * dst_stride;
        memcpy(q, p, elem_size);
    }
}

static __global__ void array_copy_2d_kernel(void* dst, const void* src,
                                        wp::vec_t<2, int> dst_strides, wp::vec_t<2, int> src_strides,
                                        wp::vec_t<2, const int*> dst_indices, wp::vec_t<2, const int*> src_indices,
                                        wp::vec_t<2, int> shape, int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int n = shape[1];
    int i = tid / n;
    int j = tid % n;
    if (i < shape[0] /*&& j < shape[1]*/)
    {
        int src_idx0 = src_indices[0] ? src_indices[0][i] : i;
        int dst_idx0 = dst_indices[0] ? dst_indices[0][i] : i;
        int src_idx1 = src_indices[1] ? src_indices[1][j] : j;
        int dst_idx1 = dst_indices[1] ? dst_indices[1][j] : j;
        const char* p = (const char*)src + src_idx0 * src_strides[0] + src_idx1 * src_strides[1];
        char* q = (char*)dst + dst_idx0 * dst_strides[0] + dst_idx1 * dst_strides[1];
        memcpy(q, p, elem_size);
    }
}

static __global__ void array_copy_3d_kernel(void* dst, const void* src,
                                        wp::vec_t<3, int> dst_strides, wp::vec_t<3, int> src_strides,
                                        wp::vec_t<3, const int*> dst_indices, wp::vec_t<3, const int*> src_indices,
                                        wp::vec_t<3, int> shape, int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int n = shape[1];
    int o = shape[2];
    int i = tid / (n * o);
    int j = tid % (n * o) / o;
    int k = tid % o;
    if (i < shape[0] && j < shape[1] /*&& k < shape[2]*/)
    {
        int src_idx0 = src_indices[0] ? src_indices[0][i] : i;
        int dst_idx0 = dst_indices[0] ? dst_indices[0][i] : i;
        int src_idx1 = src_indices[1] ? src_indices[1][j] : j;
        int dst_idx1 = dst_indices[1] ? dst_indices[1][j] : j;
        int src_idx2 = src_indices[2] ? src_indices[2][k] : k;
        int dst_idx2 = dst_indices[2] ? dst_indices[2][k] : k;
        const char* p = (const char*)src + src_idx0 * src_strides[0]
                                         + src_idx1 * src_strides[1]
                                         + src_idx2 * src_strides[2];
        char* q = (char*)dst + dst_idx0 * dst_strides[0]
                             + dst_idx1 * dst_strides[1]
                             + dst_idx2 * dst_strides[2];
        memcpy(q, p, elem_size);
    }
}

static __global__ void array_copy_4d_kernel(void* dst, const void* src,
                                        wp::vec_t<4, int> dst_strides, wp::vec_t<4, int> src_strides,
                                        wp::vec_t<4, const int*> dst_indices, wp::vec_t<4, const int*> src_indices,
                                        wp::vec_t<4, int> shape, int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int n = shape[1];
    int o = shape[2];
    int p = shape[3];
    int i = tid / (n * o * p);
    int j = tid % (n * o * p) / (o * p);
    int k = tid % (o * p) / p;
    int l = tid % p;
    if (i < shape[0] && j < shape[1] && k < shape[2] /*&& l < shape[3]*/)
    {
        int src_idx0 = src_indices[0] ? src_indices[0][i] : i;
        int dst_idx0 = dst_indices[0] ? dst_indices[0][i] : i;
        int src_idx1 = src_indices[1] ? src_indices[1][j] : j;
        int dst_idx1 = dst_indices[1] ? dst_indices[1][j] : j;
        int src_idx2 = src_indices[2] ? src_indices[2][k] : k;
        int dst_idx2 = dst_indices[2] ? dst_indices[2][k] : k;
        int src_idx3 = src_indices[3] ? src_indices[3][l] : l;
        int dst_idx3 = dst_indices[3] ? dst_indices[3][l] : l;
        const char* p = (const char*)src + src_idx0 * src_strides[0]
                                         + src_idx1 * src_strides[1]
                                         + src_idx2 * src_strides[2]
                                         + src_idx3 * src_strides[3];
        char* q = (char*)dst + dst_idx0 * dst_strides[0]
                             + dst_idx1 * dst_strides[1]
                             + dst_idx2 * dst_strides[2]
                             + dst_idx3 * dst_strides[3];
        memcpy(q, p, elem_size);
    }
}


static __global__ void array_copy_from_fabric_kernel(wp::fabricarray_t<void> src,
                                                     void* dst_data, int dst_stride, const int* dst_indices,
                                                     int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < src.size)
    {
        int dst_idx = dst_indices ? dst_indices[tid] : tid;
        void* dst_ptr = (char*)dst_data + dst_idx * dst_stride;
        const void* src_ptr = fabricarray_element_ptr(src, tid, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}

static __global__ void array_copy_from_fabric_indexed_kernel(wp::indexedfabricarray_t<void> src,
                                                             void* dst_data, int dst_stride, const int* dst_indices,
                                                             int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < src.size)
    {
        int src_index = src.indices[tid];
        int dst_idx = dst_indices ? dst_indices[tid] : tid;
        void* dst_ptr = (char*)dst_data + dst_idx * dst_stride;
        const void* src_ptr = fabricarray_element_ptr(src.fa, src_index, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}

static __global__ void array_copy_to_fabric_kernel(wp::fabricarray_t<void> dst,
                                                   const void* src_data, int src_stride, const int* src_indices,
                                                   int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dst.size)
    {
        int src_idx = src_indices ? src_indices[tid] : tid;
        const void* src_ptr = (const char*)src_data + src_idx * src_stride;
        void* dst_ptr = fabricarray_element_ptr(dst, tid, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}

static __global__ void array_copy_to_fabric_indexed_kernel(wp::indexedfabricarray_t<void> dst,
                                                           const void* src_data, int src_stride, const int* src_indices,
                                                           int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dst.size)
    {
        int src_idx = src_indices ? src_indices[tid] : tid;
        const void* src_ptr = (const char*)src_data + src_idx * src_stride;
        int dst_idx = dst.indices[tid];
        void* dst_ptr = fabricarray_element_ptr(dst.fa, dst_idx, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}


static __global__ void array_copy_fabric_to_fabric_kernel(wp::fabricarray_t<void> dst, wp::fabricarray_t<void> src, int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dst.size)
    {
        const void* src_ptr = fabricarray_element_ptr(src, tid, elem_size);
        void* dst_ptr = fabricarray_element_ptr(dst, tid, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}


static __global__ void array_copy_fabric_to_fabric_indexed_kernel(wp::indexedfabricarray_t<void> dst, wp::fabricarray_t<void> src, int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dst.size)
    {
        const void* src_ptr = fabricarray_element_ptr(src, tid, elem_size);
        int dst_index = dst.indices[tid];
        void* dst_ptr = fabricarray_element_ptr(dst.fa, dst_index, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}


static __global__ void array_copy_fabric_indexed_to_fabric_kernel(wp::fabricarray_t<void> dst, wp::indexedfabricarray_t<void> src, int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dst.size)
    {
        int src_index = src.indices[tid];
        const void* src_ptr = fabricarray_element_ptr(src.fa, src_index, elem_size);
        void* dst_ptr = fabricarray_element_ptr(dst, tid, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}


static __global__ void array_copy_fabric_indexed_to_fabric_indexed_kernel(wp::indexedfabricarray_t<void> dst, wp::indexedfabricarray_t<void> src, int elem_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dst.size)
    {
        int src_index = src.indices[tid];
        int dst_index = dst.indices[tid];
        const void* src_ptr = fabricarray_element_ptr(src.fa, src_index, elem_size);
        void* dst_ptr = fabricarray_element_ptr(dst.fa, dst_index, elem_size);
        memcpy(dst_ptr, src_ptr, elem_size);
    }
}


WP_API bool array_copy_device(void* context, void* dst, void* src, int dst_type, int src_type, int elem_size)
{
    if (!src || !dst)
        return false;

    const void* src_data = NULL;
    void* dst_data = NULL;
    int src_ndim = 0;
    int dst_ndim = 0;
    const int* src_shape = NULL;
    const int* dst_shape = NULL;
    const int* src_strides = NULL;
    const int* dst_strides = NULL;
    const int*const* src_indices = NULL;
    const int*const* dst_indices = NULL;

    const wp::fabricarray_t<void>* src_fabricarray = NULL;
    wp::fabricarray_t<void>* dst_fabricarray = NULL;

    const wp::indexedfabricarray_t<void>* src_indexedfabricarray = NULL;
    wp::indexedfabricarray_t<void>* dst_indexedfabricarray = NULL;

    const int* null_indices[wp::ARRAY_MAX_DIMS] = { NULL };

    if (src_type == wp::ARRAY_TYPE_REGULAR)
    {
        const wp::array_t<void>& src_arr = *static_cast<const wp::array_t<void>*>(src);
        src_data = src_arr.data;
        src_ndim = src_arr.ndim;
        src_shape = src_arr.shape.dims;
        src_strides = src_arr.strides;
        src_indices = null_indices;
    }
    else if (src_type == wp::ARRAY_TYPE_INDEXED)
    {
        const wp::indexedarray_t<void>& src_arr = *static_cast<const wp::indexedarray_t<void>*>(src);
        src_data = src_arr.arr.data;
        src_ndim = src_arr.arr.ndim;
        src_shape = src_arr.shape.dims;
        src_strides = src_arr.arr.strides;
        src_indices = src_arr.indices;
    }
    else if (src_type == wp::ARRAY_TYPE_FABRIC)
    {
        src_fabricarray = static_cast<const wp::fabricarray_t<void>*>(src);
        src_ndim = 1;
    }
    else if (src_type == wp::ARRAY_TYPE_FABRIC_INDEXED)
    {
        src_indexedfabricarray = static_cast<const wp::indexedfabricarray_t<void>*>(src);
        src_ndim = 1;
    }
    else
    {
        fprintf(stderr, "Warp copy error: Invalid array type (%d)\n", src_type);
        return false;
    }

    if (dst_type == wp::ARRAY_TYPE_REGULAR)
    {
        const wp::array_t<void>& dst_arr = *static_cast<const wp::array_t<void>*>(dst);
        dst_data = dst_arr.data;
        dst_ndim = dst_arr.ndim;
        dst_shape = dst_arr.shape.dims;
        dst_strides = dst_arr.strides;
        dst_indices = null_indices;
    }
    else if (dst_type == wp::ARRAY_TYPE_INDEXED)
    {
        const wp::indexedarray_t<void>& dst_arr = *static_cast<const wp::indexedarray_t<void>*>(dst);
        dst_data = dst_arr.arr.data;
        dst_ndim = dst_arr.arr.ndim;
        dst_shape = dst_arr.shape.dims;
        dst_strides = dst_arr.arr.strides;
        dst_indices = dst_arr.indices;
    }
    else if (dst_type == wp::ARRAY_TYPE_FABRIC)
    {
        dst_fabricarray = static_cast<wp::fabricarray_t<void>*>(dst);
        dst_ndim = 1;
    }
    else if (dst_type == wp::ARRAY_TYPE_FABRIC_INDEXED)
    {
        dst_indexedfabricarray = static_cast<wp::indexedfabricarray_t<void>*>(dst);
        dst_ndim = 1;
    }
    else
    {
        fprintf(stderr, "Warp copy error: Invalid array type (%d)\n", dst_type);
        return false;
    }

    if (src_ndim != dst_ndim)
    {
        fprintf(stderr, "Warp copy error: Incompatible array dimensionalities (%d and %d)\n", src_ndim, dst_ndim);
        return false;
    }

    ContextGuard guard(context);

    // handle fabric arrays
    if (dst_fabricarray)
    {
        size_t n = dst_fabricarray->size;
        if (src_fabricarray)
        {
            // copy from fabric to fabric
            if (src_fabricarray->size != n)
            {
                fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
                return false;
            }
            wp_launch_device(WP_CURRENT_CONTEXT, array_copy_fabric_to_fabric_kernel, n,
                            (*dst_fabricarray, *src_fabricarray, elem_size));
            return true;
        }
        else if (src_indexedfabricarray)
        {
            // copy from fabric indexed to fabric
            if (src_indexedfabricarray->size != n)
            {
                fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
                return false;
            }
            wp_launch_device(WP_CURRENT_CONTEXT, array_copy_fabric_indexed_to_fabric_kernel, n,
                            (*dst_fabricarray, *src_indexedfabricarray, elem_size));
            return true;
        }
        else
        {
            // copy to fabric
            if (size_t(src_shape[0]) != n)
            {
                fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
                return false;
            }
            wp_launch_device(WP_CURRENT_CONTEXT, array_copy_to_fabric_kernel, n,
                            (*dst_fabricarray, src_data, src_strides[0], src_indices[0], elem_size));
            return true;
        }
    }
    if (dst_indexedfabricarray)
    {
        size_t n = dst_indexedfabricarray->size;
        if (src_fabricarray)
        {
            // copy from fabric to fabric indexed
            if (src_fabricarray->size != n)
            {
                fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
                return false;
            }
            wp_launch_device(WP_CURRENT_CONTEXT, array_copy_fabric_to_fabric_indexed_kernel, n,
                            (*dst_indexedfabricarray, *src_fabricarray, elem_size));
            return true;
        }
        else if (src_indexedfabricarray)
        {
            // copy from fabric indexed to fabric indexed
            if (src_indexedfabricarray->size != n)
            {
                fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
                return false;
            }
            wp_launch_device(WP_CURRENT_CONTEXT, array_copy_fabric_indexed_to_fabric_indexed_kernel, n,
                            (*dst_indexedfabricarray, *src_indexedfabricarray, elem_size));
            return true;
        }
        else
        {
            // copy to fabric indexed
            if (size_t(src_shape[0]) != n)
            {
                fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
                return false;
            }
            wp_launch_device(WP_CURRENT_CONTEXT, array_copy_to_fabric_indexed_kernel, n,
                             (*dst_indexedfabricarray, src_data, src_strides[0], src_indices[0], elem_size));
            return true;
        }
    }
    else if (src_fabricarray)
    {
        // copy from fabric
        size_t n = src_fabricarray->size;
        if (size_t(dst_shape[0]) != n)
        {
            fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
            return false;
        }
        wp_launch_device(WP_CURRENT_CONTEXT, array_copy_from_fabric_kernel, n,
                         (*src_fabricarray, dst_data, dst_strides[0], dst_indices[0], elem_size));
        return true;
    }
    else if (src_indexedfabricarray)
    {
        // copy from fabric indexed
        size_t n = src_indexedfabricarray->size;
        if (size_t(dst_shape[0]) != n)
        {
            fprintf(stderr, "Warp copy error: Incompatible array sizes\n");
            return false;
        }
        wp_launch_device(WP_CURRENT_CONTEXT, array_copy_from_fabric_indexed_kernel, n,
                         (*src_indexedfabricarray, dst_data, dst_strides[0], dst_indices[0], elem_size));
        return true;
    }

    size_t n = 1;
    for (int i = 0; i < src_ndim; i++)
    {
        if (src_shape[i] != dst_shape[i])
        {
            fprintf(stderr, "Warp copy error: Incompatible array shapes\n");
            return false;
        }
        n *= src_shape[i];
    }

    switch (src_ndim)
    {
    case 1:
    {
        wp_launch_device(WP_CURRENT_CONTEXT, array_copy_1d_kernel, n, (dst_data, src_data,
                                                                   dst_strides[0], src_strides[0],
                                                                   dst_indices[0], src_indices[0],
                                                                   src_shape[0], elem_size));
        break;
    }
    case 2:
    {
        wp::vec_t<2, int> shape_v(src_shape[0], src_shape[1]);
        wp::vec_t<2, int> src_strides_v(src_strides[0], src_strides[1]);
        wp::vec_t<2, int> dst_strides_v(dst_strides[0], dst_strides[1]);
        wp::vec_t<2, const int*> src_indices_v(src_indices[0], src_indices[1]);
        wp::vec_t<2, const int*> dst_indices_v(dst_indices[0], dst_indices[1]);

        wp_launch_device(WP_CURRENT_CONTEXT, array_copy_2d_kernel, n, (dst_data, src_data,
                                                                   dst_strides_v, src_strides_v,
                                                                   dst_indices_v, src_indices_v,
                                                                   shape_v, elem_size));
        break;
    }
    case 3:
    {
        wp::vec_t<3, int> shape_v(src_shape[0], src_shape[1], src_shape[2]);
        wp::vec_t<3, int> src_strides_v(src_strides[0], src_strides[1], src_strides[2]);
        wp::vec_t<3, int> dst_strides_v(dst_strides[0], dst_strides[1], dst_strides[2]);
        wp::vec_t<3, const int*> src_indices_v(src_indices[0], src_indices[1], src_indices[2]);
        wp::vec_t<3, const int*> dst_indices_v(dst_indices[0], dst_indices[1], dst_indices[2]);

        wp_launch_device(WP_CURRENT_CONTEXT, array_copy_3d_kernel, n, (dst_data, src_data,
                                                                   dst_strides_v, src_strides_v,
                                                                   dst_indices_v, src_indices_v,
                                                                   shape_v, elem_size));
        break;
    }
    case 4:
    {
        wp::vec_t<4, int> shape_v(src_shape[0], src_shape[1], src_shape[2], src_shape[3]);
        wp::vec_t<4, int> src_strides_v(src_strides[0], src_strides[1], src_strides[2], src_strides[3]);
        wp::vec_t<4, int> dst_strides_v(dst_strides[0], dst_strides[1], dst_strides[2], dst_strides[3]);
        wp::vec_t<4, const int*> src_indices_v(src_indices[0], src_indices[1], src_indices[2], src_indices[3]);
        wp::vec_t<4, const int*> dst_indices_v(dst_indices[0], dst_indices[1], dst_indices[2], dst_indices[3]);

        wp_launch_device(WP_CURRENT_CONTEXT, array_copy_4d_kernel, n, (dst_data, src_data,
                                                                   dst_strides_v, src_strides_v,
                                                                   dst_indices_v, src_indices_v,
                                                                   shape_v, elem_size));
        break;
    }
    default:
        fprintf(stderr, "Warp copy error: invalid array dimensionality (%d)\n", src_ndim);
        return false;
    }

    return check_cuda(hipGetLastError());
}


static __global__ void array_fill_1d_kernel(void* data,
                                            int n,
                                            int stride,
                                            const int* indices,
                                            const void* value,
                                            int value_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        int idx = indices ? indices[i] : i;
        char* p = (char*)data + idx * stride;
        memcpy(p, value, value_size);
    }
}

static __global__ void array_fill_2d_kernel(void* data,
                                            wp::vec_t<2, int> shape,
                                            wp::vec_t<2, int> strides,
                                            wp::vec_t<2, const int*> indices,
                                            const void* value,
                                            int value_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int n = shape[1];
    int i = tid / n;
    int j = tid % n;
    if (i < shape[0] /*&& j < shape[1]*/)
    {
        int idx0 = indices[0] ? indices[0][i] : i;
        int idx1 = indices[1] ? indices[1][j] : j;
        char* p = (char*)data + idx0 * strides[0] + idx1 * strides[1];
        memcpy(p, value, value_size);
    }
}

static __global__ void array_fill_3d_kernel(void* data,
                                            wp::vec_t<3, int> shape,
                                            wp::vec_t<3, int> strides,
                                            wp::vec_t<3, const int*> indices,
                                            const void* value,
                                            int value_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int n = shape[1];
    int o = shape[2];
    int i = tid / (n * o);
    int j = tid % (n * o) / o;
    int k = tid % o;
    if (i < shape[0] && j < shape[1] /*&& k < shape[2]*/)
    {
        int idx0 = indices[0] ? indices[0][i] : i;
        int idx1 = indices[1] ? indices[1][j] : j;
        int idx2 = indices[2] ? indices[2][k] : k;
        char* p = (char*)data + idx0 * strides[0] + idx1 * strides[1] + idx2 * strides[2];
        memcpy(p, value, value_size);
    }
}

static __global__ void array_fill_4d_kernel(void* data,
                                            wp::vec_t<4, int> shape,
                                            wp::vec_t<4, int> strides,
                                            wp::vec_t<4, const int*> indices,
                                            const void* value,
                                            int value_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int n = shape[1];
    int o = shape[2];
    int p = shape[3];
    int i = tid / (n * o * p);
    int j = tid % (n * o * p) / (o * p);
    int k = tid % (o * p) / p;
    int l = tid % p;
    if (i < shape[0] && j < shape[1] && k < shape[2] /*&& l < shape[3]*/)
    {
        int idx0 = indices[0] ? indices[0][i] : i;
        int idx1 = indices[1] ? indices[1][j] : j;
        int idx2 = indices[2] ? indices[2][k] : k;
        int idx3 = indices[3] ? indices[3][l] : l;
        char* p = (char*)data + idx0 * strides[0] + idx1 * strides[1] + idx2 * strides[2] + idx3 * strides[3];
        memcpy(p, value, value_size);
    }
}


static __global__ void array_fill_fabric_kernel(wp::fabricarray_t<void> fa, const void* value, int value_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < fa.size)
    {
        void* dst_ptr = fabricarray_element_ptr(fa, tid, value_size);
        memcpy(dst_ptr, value, value_size);
    }
}


static __global__ void array_fill_fabric_indexed_kernel(wp::indexedfabricarray_t<void> ifa, const void* value, int value_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < ifa.size)
    {
        size_t idx = size_t(ifa.indices[tid]);
        if (idx < ifa.fa.size)
        {
            void* dst_ptr = fabricarray_element_ptr(ifa.fa, idx, value_size);
            memcpy(dst_ptr, value, value_size);
        }
    }
}


WP_API void array_fill_device(void* context, void* arr_ptr, int arr_type, const void* value_ptr, int value_size)
{
    if (!arr_ptr || !value_ptr)
        return;

    void* data = NULL;
    int ndim = 0;
    const int* shape = NULL;
    const int* strides = NULL;
    const int*const* indices = NULL;

    wp::fabricarray_t<void>* fa = NULL;
    wp::indexedfabricarray_t<void>* ifa = NULL;

    const int* null_indices[wp::ARRAY_MAX_DIMS] = { NULL };

    if (arr_type == wp::ARRAY_TYPE_REGULAR)
    {
        wp::array_t<void>& arr = *static_cast<wp::array_t<void>*>(arr_ptr);
        data = arr.data;
        ndim = arr.ndim;
        shape = arr.shape.dims;
        strides = arr.strides;
        indices = null_indices;
    }
    else if (arr_type == wp::ARRAY_TYPE_INDEXED)
    {
        wp::indexedarray_t<void>& ia = *static_cast<wp::indexedarray_t<void>*>(arr_ptr);
        data = ia.arr.data;
        ndim = ia.arr.ndim;
        shape = ia.shape.dims;
        strides = ia.arr.strides;
        indices = ia.indices;
    }
    else if (arr_type == wp::ARRAY_TYPE_FABRIC)
    {
        fa = static_cast<wp::fabricarray_t<void>*>(arr_ptr);
    }
    else if (arr_type == wp::ARRAY_TYPE_FABRIC_INDEXED)
    {
        ifa = static_cast<wp::indexedfabricarray_t<void>*>(arr_ptr);
    }
    else
    {
        fprintf(stderr, "Warp fill error: Invalid array type id %d\n", arr_type);
        return;
    }

    size_t n = 1;
    for (int i = 0; i < ndim; i++)
        n *= shape[i];

    ContextGuard guard(context);

    // copy value to device memory
    // TODO: use a persistent stream-local staging buffer to avoid allocs?
    void* value_devptr = alloc_device(WP_CURRENT_CONTEXT, value_size);
    check_cuda(hipMemcpyAsync(value_devptr, value_ptr, value_size, hipMemcpyHostToDevice, get_current_stream()));

    // handle fabric arrays
    if (fa)
    {
        wp_launch_device(WP_CURRENT_CONTEXT, array_fill_fabric_kernel, n,
                         (*fa, value_devptr, value_size));
        return;
    }
    else if (ifa)
    {
        wp_launch_device(WP_CURRENT_CONTEXT, array_fill_fabric_indexed_kernel, n,
                         (*ifa, value_devptr, value_size));
        return;
    }

    // handle regular or indexed arrays
    switch (ndim)
    {
    case 1:
    {
        wp_launch_device(WP_CURRENT_CONTEXT, array_fill_1d_kernel, n,
                         (data, shape[0], strides[0], indices[0], value_devptr, value_size));
        break;
    }
    case 2:
    {
        wp::vec_t<2, int> shape_v(shape[0], shape[1]);
        wp::vec_t<2, int> strides_v(strides[0], strides[1]);
        wp::vec_t<2, const int*> indices_v(indices[0], indices[1]);
        wp_launch_device(WP_CURRENT_CONTEXT, array_fill_2d_kernel, n,
                         (data, shape_v, strides_v, indices_v, value_devptr, value_size));
        break;
    }
    case 3:
    {
        wp::vec_t<3, int> shape_v(shape[0], shape[1], shape[2]);
        wp::vec_t<3, int> strides_v(strides[0], strides[1], strides[2]);
        wp::vec_t<3, const int*> indices_v(indices[0], indices[1], indices[2]);
        wp_launch_device(WP_CURRENT_CONTEXT, array_fill_3d_kernel, n,
                         (data, shape_v, strides_v, indices_v, value_devptr, value_size));
        break;
    }
    case 4:
    {
        wp::vec_t<4, int> shape_v(shape[0], shape[1], shape[2], shape[3]);
        wp::vec_t<4, int> strides_v(strides[0], strides[1], strides[2], strides[3]);
        wp::vec_t<4, const int*> indices_v(indices[0], indices[1], indices[2], indices[3]);
        wp_launch_device(WP_CURRENT_CONTEXT, array_fill_4d_kernel, n,
                         (data, shape_v, strides_v, indices_v, value_devptr, value_size));
        break;
    }
    default:
        fprintf(stderr, "Warp fill error: invalid array dimensionality (%d)\n", ndim);
        return;
    }

    free_device(WP_CURRENT_CONTEXT, value_devptr);
}

void array_scan_int_device(uint64_t in, uint64_t out, int len, bool inclusive)
{
    scan_device((const int*)in, (int*)out, len, inclusive);
}

void array_scan_float_device(uint64_t in, uint64_t out, int len, bool inclusive)
{
    scan_device((const float*)in, (float*)out, len, inclusive);
}

int cuda_driver_version()
{
    int version;
    if (check_cu(cuDriverGetVersion_f(&version)))
        return version;
    else
        return 0;
}

int cuda_toolkit_version()
{
    return CUDA_VERSION;
}

bool cuda_driver_is_initialized()
{
    return is_cuda_driver_initialized();
}

int nvrtc_supported_arch_count()
{
    int count;
    if (check_nvrtc(nvrtcGetNumSupportedArchs(&count)))
        return count;
    else
        return 0;
}

void nvrtc_supported_archs(int* archs)
{
    if (archs)
    {
        check_nvrtc(nvrtcGetSupportedArchs(archs));
    }
}

int cuda_device_get_count()
{
    int count = 0;
    check_cu(cuDeviceGetCount_f(&count));
    return count;
}

void* cuda_device_get_primary_context(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
    {
        DeviceInfo& device_info = g_devices[ordinal];

        // acquire the primary context if we haven't already
        if (!device_info.primary_context)
            check_cu(cuDevicePrimaryCtxRetain_f(&device_info.primary_context, device_info.device));

        return device_info.primary_context;
    }

    return NULL;
}

const char* cuda_device_get_name(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].name;
    return NULL;
}

int cuda_device_get_arch(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].arch;
    return 0;
}

void cuda_device_get_uuid(int ordinal, char uuid[16])
{
    memcpy(uuid, g_devices[ordinal].uuid.bytes, sizeof(char)*16);
}

int cuda_device_get_pci_domain_id(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].pci_domain_id;
    return -1;
}

int cuda_device_get_pci_bus_id(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].pci_bus_id;
    return -1;
}

int cuda_device_get_pci_device_id(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].pci_device_id;
    return -1;
}

int cuda_device_is_uva(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].is_uva;
    return 0;
}

int cuda_device_is_mempool_supported(int ordinal)
{
    if (ordinal >= 0 && ordinal < int(g_devices.size()))
        return g_devices[ordinal].is_mempool_supported;
    return 0;
}

int cuda_device_set_mempool_release_threshold(int ordinal, uint64_t threshold)
{
    if (ordinal < 0 || ordinal > int(g_devices.size()))
    {
        fprintf(stderr, "Invalid device ordinal %d\n", ordinal);
        return 0;
    }

    if (!g_devices[ordinal].is_mempool_supported)
        return 0;

    hipMemPool_t pool;
    if (!check_cuda(hipDeviceGetDefaultMemPool(&pool, ordinal)))
    {
        fprintf(stderr, "Warp error: Failed to get memory pool on device %d\n", ordinal);
        return 0;
    }

    if (!check_cuda(hipMemPoolSetAttribute(pool, hipMemPoolAttrReleaseThreshold, &threshold)))
    {
        fprintf(stderr, "Warp error: Failed to set memory pool attribute on device %d\n", ordinal);
        return 0;
    }

    return 1;  // success
}

uint64_t cuda_device_get_mempool_release_threshold(int ordinal)
{
    if (ordinal < 0 || ordinal > int(g_devices.size()))
    {
        fprintf(stderr, "Invalid device ordinal %d\n", ordinal);
        return 0;
    }

    if (!g_devices[ordinal].is_mempool_supported)
        return 0;

    hipMemPool_t pool;
    if (!check_cuda(hipDeviceGetDefaultMemPool(&pool, ordinal)))
    {
        fprintf(stderr, "Warp error: Failed to get memory pool on device %d\n", ordinal);
        return 0;
    }

    uint64_t threshold = 0;
    if (!check_cuda(hipMemPoolGetAttribute(pool, hipMemPoolAttrReleaseThreshold, &threshold)))
    {
        fprintf(stderr, "Warp error: Failed to get memory pool release threshold on device %d\n", ordinal);
        return 0;
    }

    return threshold;
}

void cuda_device_get_memory_info(int ordinal, size_t* free_mem, size_t* total_mem)
{
    // use temporary storage if user didn't specify pointers
    size_t tmp_free_mem, tmp_total_mem;

    if (free_mem)
        *free_mem = 0;
    else
        free_mem = &tmp_free_mem;

    if (total_mem)
        *total_mem = 0;
    else
        total_mem = &tmp_total_mem;

    if (ordinal >= 0 && ordinal < int(g_devices.size()))
    {
        if (g_devices[ordinal].primary_context)
        {
            ContextGuard guard(g_devices[ordinal].primary_context, true);
            check_cu(cuMemGetInfo_f(free_mem, total_mem));
        }
        else
        {
            // if we haven't acquired the primary context yet, acquire it temporarily
            hipCtx_t primary_context = NULL;
            check_cu(cuDevicePrimaryCtxRetain_f(&primary_context, g_devices[ordinal].device));
            {
                ContextGuard guard(primary_context, true);
                check_cu(cuMemGetInfo_f(free_mem, total_mem));
            }
            check_cu(cuDevicePrimaryCtxRelease_f(g_devices[ordinal].device));
        }
    }
}


void* cuda_context_get_current()
{
    return get_current_context();
}

void cuda_context_set_current(void* context)
{
    hipCtx_t ctx = static_cast<hipCtx_t>(context);
    hipCtx_t prev_ctx = NULL;
    check_cu(cuCtxGetCurrent_f(&prev_ctx));
    if (ctx != prev_ctx)
    {
        check_cu(cuCtxSetCurrent_f(ctx));
    }
}

void cuda_context_push_current(void* context)
{
    check_cu(cuCtxPushCurrent_f(static_cast<hipCtx_t>(context)));
}

void cuda_context_pop_current()
{
    hipCtx_t context;
    check_cu(cuCtxPopCurrent_f(&context));
}

void* cuda_context_create(int device_ordinal)
{
    hipCtx_t ctx = NULL;
    hipDevice_t device;
    if (check_cu(cuDeviceGet_f(&device, device_ordinal)))
        check_cu(cuCtxCreate_f(&ctx, 0, device));
    return ctx;
}

void cuda_context_destroy(void* context)
{
    if (context)
    {
        hipCtx_t ctx = static_cast<hipCtx_t>(context);

        // ensure this is not the current context
        if (ctx == cuda_context_get_current())
            cuda_context_set_current(NULL);

        // release the cached info about this context
        ContextInfo* info = get_context_info(ctx);
        if (info)
        {
            if (info->stream)
                check_cu(cuStreamDestroy_f(info->stream));
            
            g_contexts.erase(ctx);
        }

        check_cu(cuCtxDestroy_f(ctx));
    }
}

void cuda_context_synchronize(void* context)
{
    ContextGuard guard(context);

    check_cu(cuCtxSynchronize_f());

    if (free_deferred_allocs(context ? context : get_current_context()) > 0)
    {
        // ensure deferred asynchronous deallocations complete
        check_cu(cuCtxSynchronize_f());
    }

    // check_cuda(hipDeviceGraphMemTrim(cuda_context_get_device_ordinal(context)));
}

uint64_t cuda_context_check(void* context)
{
    ContextGuard guard(context);

    // check errors before syncing
    hipError_t e = hipGetLastError();
    check_cuda(e);

    hipStreamCaptureStatus status = hipStreamCaptureStatusNone;
    check_cuda(hipStreamIsCapturing(get_current_stream(), &status));
    
    // synchronize if the stream is not capturing
    if (status == hipStreamCaptureStatusNone)
    {
        check_cuda(hipDeviceSynchronize());
        e = hipGetLastError();
    }

    return static_cast<uint64_t>(e);
}


int cuda_context_get_device_ordinal(void* context)
{
    ContextInfo* info = get_context_info(static_cast<hipCtx_t>(context));
    return info && info->device_info ? info->device_info->ordinal : -1;
}

int cuda_context_is_primary(void* context)
{
    hipCtx_t ctx = static_cast<hipCtx_t>(context);
    ContextInfo* context_info = get_context_info(ctx);
    if (!context_info)
    {
        fprintf(stderr, "Warp error: Failed to get context info\n");
        return 0;
    }

    // if the device primary context is known, check if it matches the given context
    DeviceInfo* device_info = context_info->device_info;
    if (device_info->primary_context)
        return int(ctx == device_info->primary_context);

    // there is no CUDA API to check if a context is primary, but we can temporarily
    // acquire the device's primary context to check the pointer
    hipCtx_t primary_ctx;
    if (check_cu(cuDevicePrimaryCtxRetain_f(&primary_ctx, device_info->device)))
    {
        check_cu(cuDevicePrimaryCtxRelease_f(device_info->device));
        return int(ctx == primary_ctx);
    }

    return 0;
}

void* cuda_context_get_stream(void* context)
{
    ContextInfo* info = get_context_info(static_cast<hipCtx_t>(context));
    if (info)
    {
        return info->stream;
    }
    return NULL;
}

void cuda_context_set_stream(void* context, void* stream, int sync)
{
    ContextInfo* context_info = get_context_info(static_cast<hipCtx_t>(context));
    if (context_info)
    {
        hipStream_t new_stream = static_cast<hipStream_t>(stream);

        // check whether we should sync with the previous stream on this device
        if (sync)
        {
            hipStream_t old_stream = context_info->stream;
            StreamInfo* old_stream_info = get_stream_info(old_stream);
            if (old_stream_info)
            {
                hipEvent_t cached_event = old_stream_info->cached_event;
                check_cu(cuEventRecord_f(cached_event, old_stream));
                check_cu(cuStreamWaitEvent_f(new_stream, cached_event, CU_EVENT_WAIT_DEFAULT));
            }
        }

        context_info->stream = new_stream;
    }
}


int cuda_is_peer_access_supported(int target_ordinal, int peer_ordinal)
{
    int num_devices = int(g_devices.size());

    if (target_ordinal < 0 || target_ordinal > num_devices)
    {
        fprintf(stderr, "Warp error: Invalid target device ordinal %d\n", target_ordinal);
        return 0;
    }

    if (peer_ordinal < 0 || peer_ordinal > num_devices)
    {
        fprintf(stderr, "Warp error: Invalid peer device ordinal %d\n", peer_ordinal);
        return 0;
    }

    if (target_ordinal == peer_ordinal)
        return 1;

    int can_access = 0;
    check_cuda(hipDeviceCanAccessPeer(&can_access, peer_ordinal, target_ordinal));

    return can_access;
}

int cuda_is_peer_access_enabled(void* target_context, void* peer_context)
{
    if (!target_context || !peer_context)
    {
        fprintf(stderr, "Warp error: invalid CUDA context\n");
        return 0;
    }

    if (target_context == peer_context)
        return 1;

    int target_ordinal = cuda_context_get_device_ordinal(target_context);
    int peer_ordinal = cuda_context_get_device_ordinal(peer_context);

    // check if peer access is supported
    int can_access = 0;
    check_cuda(hipDeviceCanAccessPeer(&can_access, peer_ordinal, target_ordinal));
    if (!can_access)
        return 0;

    // There is no CUDA API to query if peer access is enabled, but we can try to enable it and check the result.

    ContextGuard guard(peer_context, true);

    hipCtx_t target_ctx = static_cast<hipCtx_t>(target_context);

    hipError_t result = cuCtxEnablePeerAccess_f(target_ctx, 0);
    if (result == hipErrorPeerAccessAlreadyEnabled)
    {
        return 1;
    }
    else if (result == hipSuccess)
    {
        // undo enablement
        check_cu(cuCtxDisablePeerAccess_f(target_ctx));
        return 0;
    }
    else
    {
        // report error
        check_cu(result);
        return 0;
    }
}

int cuda_set_peer_access_enabled(void* target_context, void* peer_context, int enable)
{
    if (!target_context || !peer_context)
    {
        fprintf(stderr, "Warp error: invalid CUDA context\n");
        return 0;
    }

    if (target_context == peer_context)
        return 1;  // no-op
        
    int target_ordinal = cuda_context_get_device_ordinal(target_context);
    int peer_ordinal = cuda_context_get_device_ordinal(peer_context);

    // check if peer access is supported
    int can_access = 0;
    check_cuda(hipDeviceCanAccessPeer(&can_access, peer_ordinal, target_ordinal));
    if (!can_access)
    {
        // failure if enabling, success if disabling
        if (enable)
        {
            fprintf(stderr, "Warp error: device %d cannot access device %d\n", peer_ordinal, target_ordinal);
            return 0;
        }
        else
            return 1;
    }

    ContextGuard guard(peer_context, true);

    hipCtx_t target_ctx = static_cast<hipCtx_t>(target_context);

    if (enable)
    {
        hipError_t status = cuCtxEnablePeerAccess_f(target_ctx, 0);
        if (status != hipSuccess && status != hipErrorPeerAccessAlreadyEnabled)
        {
            check_cu(status);
            fprintf(stderr, "Warp error: failed to enable peer access from device %d to device %d\n", peer_ordinal, target_ordinal);
            return 0;
        }
    }
    else
    {
        hipError_t status = cuCtxDisablePeerAccess_f(target_ctx);
        if (status != hipSuccess && status != hipErrorPeerAccessNotEnabled)
        {
            check_cu(status);
            fprintf(stderr, "Warp error: failed to disable peer access from device %d to device %d\n", peer_ordinal, target_ordinal);
            return 0;
        }
    }

    return 1;  // success
}

int cuda_is_mempool_access_enabled(int target_ordinal, int peer_ordinal)
{
    int num_devices = int(g_devices.size());

    if (target_ordinal < 0 || target_ordinal > num_devices)
    {
        fprintf(stderr, "Warp error: Invalid device ordinal %d\n", target_ordinal);
        return 0;
    }

    if (peer_ordinal < 0 || peer_ordinal > num_devices)
    {
        fprintf(stderr, "Warp error: Invalid peer device ordinal %d\n", peer_ordinal);
        return 0;
    }

    if (target_ordinal == peer_ordinal)
        return 1;

    hipMemPool_t pool;
    if (!check_cuda(hipDeviceGetDefaultMemPool(&pool, target_ordinal)))
    {
        fprintf(stderr, "Warp error: Failed to get memory pool of device %d\n", target_ordinal);
        return 0;
    }

    hipMemAccessFlags flags = hipMemAccessFlagsProtNone;
    hipMemLocation location;
    location.id = peer_ordinal;
    location.type = hipMemLocationTypeDevice;
    if (check_cuda(hipMemPoolGetAccess(&flags, pool, &location)))
        return int(flags != hipMemAccessFlagsProtNone);

    return 0;
}

int cuda_set_mempool_access_enabled(int target_ordinal, int peer_ordinal, int enable)
{
    int num_devices = int(g_devices.size());

    if (target_ordinal < 0 || target_ordinal > num_devices)
    {
        fprintf(stderr, "Warp error: Invalid device ordinal %d\n", target_ordinal);
        return 0;
    }

    if (peer_ordinal < 0 || peer_ordinal > num_devices)
    {
        fprintf(stderr, "Warp error: Invalid peer device ordinal %d\n", peer_ordinal);
        return 0;
    }

    if (target_ordinal == peer_ordinal)
        return 1;  // no-op

    // get the memory pool
    hipMemPool_t pool;
    if (!check_cuda(hipDeviceGetDefaultMemPool(&pool, target_ordinal)))
    {
        fprintf(stderr, "Warp error: Failed to get memory pool of device %d\n", target_ordinal);
        return 0;
    }

    hipMemAccessDesc desc;
    desc.location.type = hipMemLocationTypeDevice;
    desc.location.id = peer_ordinal;

    // only hipMemAccessFlagsProtReadWrite and hipMemAccessFlagsProtNone are supported
    if (enable)
        desc.flags = hipMemAccessFlagsProtReadWrite;
    else
        desc.flags = hipMemAccessFlagsProtNone;

    if (!check_cuda(hipMemPoolSetAccess(pool, &desc, 1)))
    {
        fprintf(stderr, "Warp error: Failed to set mempool access from device %d to device %d\n", peer_ordinal, target_ordinal);
        return 0;
    }

    return 1;  // success
}


void* cuda_stream_create(void* context)
{
    ContextGuard guard(context, true);

    hipStream_t stream;
    if (check_cu(cuStreamCreate_f(&stream, hipStreamDefault)))
    {
        cuda_stream_register(WP_CURRENT_CONTEXT, stream);
        return stream;
    }
    else
        return NULL;
}

void cuda_stream_destroy(void* context, void* stream)
{
    if (!stream)
        return;

    cuda_stream_unregister(context, stream);

    check_cu(cuStreamDestroy_f(static_cast<hipStream_t>(stream)));
}

void cuda_stream_register(void* context, void* stream)
{
    if (!stream)
        return;

    ContextGuard guard(context);

    // populate stream info
    StreamInfo& stream_info = g_streams[static_cast<hipStream_t>(stream)];
    check_cu(cuEventCreate_f(&stream_info.cached_event, hipEventDisableTiming));
}

void cuda_stream_unregister(void* context, void* stream)
{
    if (!stream)
        return;

    hipStream_t hip_stream = static_cast<hipStream_t>(stream);
    
    StreamInfo* stream_info = get_stream_info(hip_stream);
    if (stream_info)
    {
        // release stream info
        check_cu(cuEventDestroy_f(stream_info->cached_event));
        g_streams.erase(hip_stream);
    }

    // make sure we don't leave dangling references to this stream
    ContextInfo* context_info = get_context_info(context);
    if (context_info)
    {
        if (hip_stream == context_info->stream)
            context_info->stream = NULL;
    }
}

void* cuda_stream_get_current()
{
    return get_current_stream();
}

void cuda_stream_synchronize(void* stream)
{
    check_cu(cuStreamSynchronize_f(static_cast<hipStream_t>(stream)));
}

void cuda_stream_wait_event(void* stream, void* event)
{
    check_cu(cuStreamWaitEvent_f(static_cast<hipStream_t>(stream), static_cast<hipEvent_t>(event), 0));
}

void cuda_stream_wait_stream(void* stream, void* other_stream, void* event)
{
    check_cu(cuEventRecord_f(static_cast<hipEvent_t>(event), static_cast<hipStream_t>(other_stream)));
    check_cu(cuStreamWaitEvent_f(static_cast<hipStream_t>(stream), static_cast<hipEvent_t>(event), 0));
}

int cuda_stream_is_capturing(void* stream)
{
    hipStreamCaptureStatus status = hipStreamCaptureStatusNone;
    check_cuda(hipStreamIsCapturing(static_cast<hipStream_t>(stream), &status));
    
    return int(status != hipStreamCaptureStatusNone);
}

uint64_t cuda_stream_get_capture_id(void* stream)
{
    return get_capture_id(static_cast<hipStream_t>(stream));
}

void* cuda_event_create(void* context, unsigned flags)
{
    ContextGuard guard(context, true);

    hipEvent_t event;
    if (check_cu(cuEventCreate_f(&event, flags)))
        return event;
    else
        return NULL;
}

void cuda_event_destroy(void* event)
{
    check_cu(cuEventDestroy_f(static_cast<hipEvent_t>(event)));
}

void cuda_event_record(void* event, void* stream)
{
    check_cu(cuEventRecord_f(static_cast<hipEvent_t>(event), static_cast<hipStream_t>(stream)));
}

void cuda_event_synchronize(void* event)
{
    check_cu(cuEventSynchronize_f(static_cast<hipEvent_t>(event)));
}

float cuda_event_elapsed_time(void* start_event, void* end_event)
{
    float elapsed = 0.0f;
    hipEvent_t start = static_cast<hipEvent_t>(start_event);
    hipEvent_t end = static_cast<hipEvent_t>(end_event);
    check_cuda(hipEventElapsedTime(&elapsed, start, end));
    return elapsed;
}

bool cuda_graph_begin_capture(void* context, void* stream, int external)
{
    ContextGuard guard(context);

    hipStream_t hip_stream = static_cast<hipStream_t>(stream);
    StreamInfo* stream_info = get_stream_info(hip_stream);
    if (!stream_info)
    {
        wp::set_error_string("Warp error: unknown stream");
        return false;
    }

    if (external)
    {
        // if it's an external capture, make sure it's already active so we can get the capture id
        hipStreamCaptureStatus status = hipStreamCaptureStatusNone;
        if (!check_cuda(hipStreamIsCapturing(hip_stream, &status)))
            return false;
        if (status != hipStreamCaptureStatusActive)
        {
            wp::set_error_string("Warp error: stream is not capturing");
            return false;
        }
    }
    else
    {
        // start the capture
        if (!check_cuda(hipStreamBeginCapture(hip_stream, hipStreamCaptureModeGlobal)))
            return false;
    }

    uint64_t capture_id = get_capture_id(hip_stream);

    CaptureInfo* capture = new CaptureInfo();
    capture->stream = hip_stream;
    capture->id = capture_id;
    capture->external = bool(external);

    // update stream info
    stream_info->capture = capture;

    // add to known captures
    g_captures[capture_id] = capture;

    return true;
}

bool cuda_graph_end_capture(void* context, void* stream, void** graph_ret)
{
    ContextGuard guard(context);

    // check if this is a known stream
    hipStream_t hip_stream = static_cast<hipStream_t>(stream);
    StreamInfo* stream_info = get_stream_info(hip_stream);
    if (!stream_info)
    {
        wp::set_error_string("Warp error: unknown capture stream");
        return false;
    }

    // check if this stream was used to start a capture
    CaptureInfo* capture = stream_info->capture;
    if (!capture)
    {
        wp::set_error_string("Warp error: stream has no capture started");
        return false;
    }

    // get capture info
    bool external = capture->external;
    uint64_t capture_id = capture->id;

    // clear capture info
    stream_info->capture = NULL;
    g_captures.erase(capture_id);
    delete capture;

    // a lambda to clean up on exit in case of error
    auto clean_up = [hip_stream, capture_id, external]()
    {
        // unreference outstanding graph allocs so that they will be released with the user reference
        for (auto it = g_graph_allocs.begin(); it != g_graph_allocs.end(); ++it)
        {
            GraphAllocInfo& alloc_info = it->second;
            if (alloc_info.capture_id == capture_id)
                alloc_info.graph_destroyed = true;
        }

        // make sure we terminate the capture
        if (!external)
        {
            hipGraph_t graph = NULL;
            hipStreamEndCapture(hip_stream, &graph);
            hipGetLastError();
        }
    };

    // get captured graph without ending the capture in case it is external
    hipGraph_t graph = get_capture_graph(hip_stream);
    if (!graph)
    {
        clean_up();
        return false;
    }
    
    // ensure that all forked streams are joined to the main capture stream by manually
    // adding outstanding capture dependencies gathered from the graph leaf nodes
    std::vector<hipGraphNode_t> stream_dependencies;
    std::vector<hipGraphNode_t> leaf_nodes;
    if (get_capture_dependencies(hip_stream, stream_dependencies) && get_graph_leaf_nodes(graph, leaf_nodes))
    {
        // compute set difference to get unjoined dependencies
        std::vector<hipGraphNode_t> unjoined_dependencies;
        std::sort(stream_dependencies.begin(), stream_dependencies.end());
        std::sort(leaf_nodes.begin(), leaf_nodes.end());
        std::set_difference(leaf_nodes.begin(), leaf_nodes.end(),
                            stream_dependencies.begin(), stream_dependencies.end(),
                            std::back_inserter(unjoined_dependencies));
        if (!unjoined_dependencies.empty())
        {
            check_cu(cuStreamUpdateCaptureDependencies_f(hip_stream, unjoined_dependencies.data(), unjoined_dependencies.size(),
                                                         hipStreamAddCaptureDependencies));
            // ensure graph is still valid
            if (get_capture_graph(hip_stream) != graph)
            {
                clean_up();
                return false;
            }
        }
    }

    // check if this graph has unfreed allocations, which require special handling
    std::vector<void*> unfreed_allocs;
    for (auto it = g_graph_allocs.begin(); it != g_graph_allocs.end(); ++it)
    {
        GraphAllocInfo& alloc_info = it->second;
        if (alloc_info.capture_id == capture_id)
            unfreed_allocs.push_back(it->first);
    }

    if (!unfreed_allocs.empty())
    {
        // Create a user object that will notify us when the instantiated graph is destroyed.
        // This works for external captures also, since we wouldn't otherwise know when
        // the externally-created graph instance gets deleted.
        // This callback is guaranteed to arrive after the graph has finished executing on the device,
        // not necessarily when hipGraphExecDestroy() is called.
        GraphInfo* graph_info = new GraphInfo;
        graph_info->unfreed_allocs = unfreed_allocs;
        hipUserObject_t user_object;
        check_cuda(hipUserObjectCreate(&user_object, graph_info, on_graph_destroy, 1, hipUserObjectNoDestructorSync));
        check_cuda(hipGraphRetainUserObject(graph, user_object, 1, hipGraphUserObjectMove));

        // ensure graph is still valid
        if (get_capture_graph(hip_stream) != graph)
        {
            clean_up();
            return false;
        }
    }

    // for external captures, we don't instantiate the graph ourselves, so we're done
    if (external)
        return true;

    hipGraphExec_t graph_exec = NULL;

    // end the capture
    if (!check_cuda(hipStreamEndCapture(hip_stream, &graph)))
        return false;

    // enable to create debug GraphVis visualization of graph
    // hipGraphDebugDotPrint(graph, "graph.dot", hipGraphDebugDotFlagsVerbose);
    
    // can use after CUDA 11.4 to permit graphs to capture hipMallocAsync() operations
    if (!check_cuda(hipGraphInstantiateWithFlags(&graph_exec, graph, hipGraphInstantiateFlagAutoFreeOnLaunch)))
        return false;

    // free source graph
    check_cuda(hipGraphDestroy(graph));

    // process deferred free list if no more captures are ongoing
    if (g_captures.empty())
        free_deferred_allocs();

    if (graph_ret)
        *graph_ret = graph_exec;

    return true;
}

bool cuda_graph_launch(void* graph_exec, void* stream)
{
    // TODO: allow naming graphs?
    begin_cuda_range(WP_TIMING_GRAPH, stream, get_stream_context(stream), "graph");

    bool result = check_cuda(hipGraphLaunch((hipGraphExec_t)graph_exec, (hipStream_t)stream));

    end_cuda_range(WP_TIMING_GRAPH, stream);

    return result;
}

bool cuda_graph_destroy(void* context, void* graph_exec)
{
    ContextGuard guard(context);

    return check_cuda(hipGraphExecDestroy((hipGraphExec_t)graph_exec));
}

size_t cuda_compile_program(const char* cuda_src, int arch, const char* include_dir, bool debug, bool verbose, bool verify_fp, bool fast_math, const char* output_path)
{
    // use file extension to determine whether to output PTX or CUBIN
    const char* output_ext = strrchr(output_path, '.');
    bool use_ptx = output_ext && strcmp(output_ext + 1, "ptx") == 0;

    // check include dir path len (path + option)
    const int max_path = 4096 + 16;
    if (strlen(include_dir) > max_path)
    {
        fprintf(stderr, "Warp error: Include path too long\n");
        return size_t(-1);
    }

    char include_opt[max_path];
    strcpy(include_opt, "--include-path=");
    strcat(include_opt, include_dir);

    const int max_arch = 128;
    char arch_opt[max_arch];

    if (use_ptx)
        snprintf(arch_opt, max_arch, "--gpu-architecture=compute_%d", arch);
    else
        snprintf(arch_opt, max_arch, "--gpu-architecture=sm_%d", arch);

    std::vector<const char*> opts;
    opts.push_back(arch_opt);
    opts.push_back(include_opt);
    opts.push_back("--std=c++17");
    
    if (debug)
    {
        opts.push_back("--define-macro=_DEBUG");
        opts.push_back("--generate-line-info");
        // disabling since it causes issues with `Unresolved extern function 'cudaGetParameterBufferV2'
        //opts.push_back("--device-debug");
    }
    else
        opts.push_back("--define-macro=NDEBUG");

    if (verify_fp)
        opts.push_back("--define-macro=WP_VERIFY_FP");
    else
        opts.push_back("--undefine-macro=WP_VERIFY_FP");
    
    if (fast_math)
        opts.push_back("--use_fast_math");


    hiprtcProgram prog;
    hiprtcResult res;

    res = hiprtcCreateProgram(
        &prog,         // prog
        cuda_src,      // buffer
        NULL,          // name
        0,             // numHeaders
        NULL,          // headers
        NULL);         // includeNames

    if (!check_nvrtc(res))
        return size_t(res);

    res = hiprtcCompileProgram(prog, int(opts.size()), opts.data());

    if (!check_nvrtc(res) || verbose)
    {
        // get program log
        size_t log_size;
        if (check_nvrtc(hiprtcGetProgramLogSize(prog, &log_size)))
        {
            std::vector<char> log(log_size);
            if (check_nvrtc(hiprtcGetProgramLog(prog, log.data())))
            {
                // todo: figure out better way to return this to python
                if (res != HIPRTC_SUCCESS)
                    fprintf(stderr, "%s", log.data());
                else
                    fprintf(stdout, "%s", log.data());
            }
        }

        if (res != HIPRTC_SUCCESS)
        {
            hiprtcDestroyProgram(&prog);
            return size_t(res);
        }
    }

    hiprtcResult (*get_output_size)(hiprtcProgram, size_t*);
    hiprtcResult (*get_output_data)(hiprtcProgram, char*);
    const char* output_mode;
    if (use_ptx)
    {
        get_output_size = hiprtcGetCodeSize;
        get_output_data = hiprtcGetCode;
        output_mode = "wt";
    }
    else
    {
        get_output_size = hiprtcGetBitcodeSize;
        get_output_data = hiprtcGetBitcode;
        output_mode = "wb";
    }

    // save output
    size_t output_size;
    res = get_output_size(prog, &output_size);
    if (check_nvrtc(res))
    {
        std::vector<char> output(output_size);
        res = get_output_data(prog, output.data());
        if (check_nvrtc(res))
        {
            FILE* file = fopen(output_path, output_mode);
            if (file)
            {
                if (fwrite(output.data(), 1, output_size, file) != output_size)
                {
                    fprintf(stderr, "Warp error: Failed to write output file '%s'\n", output_path);
                    res = hiprtcResult(-1);
                }
                fclose(file);
            }
            else
            {
                fprintf(stderr, "Warp error: Failed to open output file '%s'\n", output_path);
                res = hiprtcResult(-1);
            }
        }
    }

    check_nvrtc(hiprtcDestroyProgram(&prog));

    return res;
}

void* cuda_load_module(void* context, const char* path)
{
    ContextGuard guard(context);

    // use file extension to determine whether to load PTX or CUBIN
    const char* input_ext = strrchr(path, '.');
    bool load_ptx = input_ext && strcmp(input_ext + 1, "ptx") == 0;

    std::vector<char> input;

    FILE* file = fopen(path, "rb");
    if (file)
    {
        fseek(file, 0, SEEK_END);
        size_t length = ftell(file);
        fseek(file, 0, SEEK_SET);

        input.resize(length + 1);
        if (fread(input.data(), 1, length, file) != length)
        {
            fprintf(stderr, "Warp error: Failed to read input file '%s'\n", path);
            fclose(file);
            return NULL;
        }
        fclose(file);

        input[length] = '\0';
    }
    else
    {
        fprintf(stderr, "Warp error: Failed to open input file '%s'\n", path);
        return NULL;
    }

    int driver_cuda_version = 0;
    hipModule_t module = NULL;

    if (load_ptx)
    {
        if (check_cu(cuDriverGetVersion_f(&driver_cuda_version)) && driver_cuda_version >= CUDA_VERSION)
        {
            // let the driver compile the PTX

            hipJitOption options[2];
            void *option_vals[2];
            char error_log[8192] = "";
            unsigned int log_size = 8192;
            // Set up loader options
            // Pass a buffer for error message
            options[0] = hipJitOptionErrorLogBuffer;
            option_vals[0] = (void*)error_log;
            // Pass the size of the error buffer
            options[1] = hipJitOptionErrorLogBufferSizeBytes;
            option_vals[1] = (void*)(size_t)log_size;

            if (!check_cu(cuModuleLoadDataEx_f(&module, input.data(), 2, options, option_vals)))
            {
                fprintf(stderr, "Warp error: Loading PTX module failed\n");
                // print error log if not empty
                if (*error_log)
                    fprintf(stderr, "PTX loader error:\n%s\n", error_log);
                return NULL;
            }
        }
        else
        {
            // manually compile the PTX and load as CUBIN

            ContextInfo* context_info = get_context_info(static_cast<hipCtx_t>(context));
            if (!context_info || !context_info->device_info)
            {
                fprintf(stderr, "Warp error: Failed to determine target architecture\n");
                return NULL;
            }

            int arch = context_info->device_info->arch;

            char arch_opt[128];
            sprintf(arch_opt, "--gpu-name=sm_%d", arch);

            const char* compiler_options[] = { arch_opt };

            nvPTXCompilerHandle compiler = NULL;
            if (!check_nvptx(nvPTXCompilerCreate(&compiler, input.size(), input.data())))
                return NULL;

            if (!check_nvptx(nvPTXCompilerCompile(compiler, sizeof(compiler_options) / sizeof(*compiler_options), compiler_options)))
                return NULL;

            size_t cubin_size = 0;
            if (!check_nvptx(nvPTXCompilerGetCompiledProgramSize(compiler, &cubin_size)))
                return NULL;

            std::vector<char> cubin(cubin_size);
            if (!check_nvptx(nvPTXCompilerGetCompiledProgram(compiler, cubin.data())))
                return NULL;

            check_nvptx(nvPTXCompilerDestroy(&compiler));

            if (!check_cu(cuModuleLoadDataEx_f(&module, cubin.data(), 0, NULL, NULL)))
            {
                fprintf(stderr, "Warp CUDA error: Loading module failed\n");
                return NULL;
            }
        }
    }
    else
    {
        // load CUBIN
        if (!check_cu(cuModuleLoadDataEx_f(&module, input.data(), 0, NULL, NULL)))
        {
            fprintf(stderr, "Warp CUDA error: Loading module failed\n");
            return NULL;
        }
    }

    return module;
}

void cuda_unload_module(void* context, void* module)
{
    ContextGuard guard(context);

    check_cu(cuModuleUnload_f((hipModule_t)module));
}

void* cuda_get_kernel(void* context, void* module, const char* name)
{
    ContextGuard guard(context);

    hipFunction_t kernel = NULL;
    if (!check_cu(cuModuleGetFunction_f(&kernel, (hipModule_t)module, name)))
    {
        fprintf(stderr, "Warp CUDA error: Failed to lookup kernel function %s in module\n", name);
        return NULL;
    }

    g_kernel_names[kernel] = name;

    return kernel;
}

size_t cuda_launch_kernel(void* context, void* kernel, size_t dim, int max_blocks, void** args, void* stream)
{
    ContextGuard guard(context);

    const int block_dim = 256;
    // CUDA specs up to compute capability 9.0 says the max x-dim grid is 2**31-1, so
    // grid_dim is fine as an int for the near future
    int grid_dim = (dim + block_dim - 1)/block_dim;

    if (max_blocks <= 0) {
        max_blocks = 2147483647;
    }

    if (grid_dim < 0)
    {
#if defined(_DEBUG)
        fprintf(stderr, "Warp warning: Overflow in grid dimensions detected for %zu total elements and 256 threads "
                "per block.\n    Setting block count to %d.\n", dim, max_blocks);
#endif
        grid_dim =  max_blocks;
    }
    else 
    {
        if (grid_dim > max_blocks)
        {
            grid_dim = max_blocks;
        }
    }

    begin_cuda_range(WP_TIMING_KERNEL, stream, context, get_cuda_kernel_name(kernel));

    hipError_t res = cuLaunchKernel_f(
        (hipFunction_t)kernel,
        grid_dim, 1, 1,
        block_dim, 1, 1,
        0, static_cast<hipStream_t>(stream),
        args,
        0);

    check_cu(res);

    end_cuda_range(WP_TIMING_KERNEL, stream);

    return res;
}

void cuda_graphics_map(void* context, void* resource)
{
    ContextGuard guard(context);

    check_cu(cuGraphicsMapResources_f(1, (hipGraphicsResource_t*)resource, get_current_stream()));
}

void cuda_graphics_unmap(void* context, void* resource)
{
    ContextGuard guard(context);

    check_cu(cuGraphicsUnmapResources_f(1, (hipGraphicsResource_t*)resource, get_current_stream()));
}

void cuda_graphics_device_ptr_and_size(void* context, void* resource, uint64_t* ptr, size_t* size)
{
    ContextGuard guard(context);

    hipDeviceptr_t device_ptr;
    size_t bytes;
    check_cu(cuGraphicsResourceGetMappedPointer_f(&device_ptr, &bytes, *(hipGraphicsResource_t*)resource));

    *ptr = device_ptr;
    *size = bytes;
}

void* cuda_graphics_register_gl_buffer(void* context, uint32_t gl_buffer, unsigned int flags)
{
    ContextGuard guard(context);

    hipGraphicsResource_t *resource = new hipGraphicsResource_t;
    bool success = check_cu(cuGraphicsGLRegisterBuffer_f(resource, gl_buffer, flags));
    if (!success)
    {
        delete resource;
        return NULL;
    }

    return resource;
}

void cuda_graphics_unregister_resource(void* context, void* resource)
{
    ContextGuard guard(context);

    hipGraphicsResource_t *res = (hipGraphicsResource_t*)resource;
    check_cu(cuGraphicsUnregisterResource_f(*res));
    delete res;
}

void cuda_timing_begin(int flags)
{
    g_cuda_timing_state = new CudaTimingState(flags, g_cuda_timing_state);
}

int cuda_timing_get_result_count()
{
    if (g_cuda_timing_state)
        return int(g_cuda_timing_state->ranges.size());
    return 0;
}

void cuda_timing_end(timing_result_t* results, int size)
{
    if (!g_cuda_timing_state)
        return;

    // number of results to write to the user buffer
    int count = std::min(cuda_timing_get_result_count(), size);

    // compute timings and write results
    for (int i = 0; i < count; i++)
    {
        const CudaTimingRange& range = g_cuda_timing_state->ranges[i];
        timing_result_t& result = results[i];
        result.context = range.context;
        result.name = range.name;
        result.flag = range.flag;
        check_cuda(hipEventElapsedTime(&result.elapsed, range.start, range.end));
    }

    // release events
    for (CudaTimingRange& range : g_cuda_timing_state->ranges)
    {
        check_cu(cuEventDestroy_f(range.start));
        check_cu(cuEventDestroy_f(range.end));
    }

    // restore previous state
    CudaTimingState* parent_state = g_cuda_timing_state->parent;
    delete g_cuda_timing_state;
    g_cuda_timing_state = parent_state;
}


// impl. files
#include "bvh.cu"
#include "mesh.cu"
#include "sort.cu"
#include "hashgrid.cu"
#include "reduce.cu"
#include "runlength_encode.cu"
#include "scan.cu"
#include "marching.cu"
#include "sparse.cu"
#include "volume.cu"
#include "volume_builder.cu"
#if WP_ENABLE_CUTLASS
    #include "cutlass_gemm.cu"
#endif

//#include "spline.inl"
//#include "volume.inl"
