#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_util.h"
#include "warp.h"

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>

namespace
{

// Combined row+column value that can be radix-sorted with CUB
using BsrRowCol = uint64_t;

static constexpr BsrRowCol PRUNED_ROWCOL = ~BsrRowCol(0);

CUDA_CALLABLE BsrRowCol bsr_combine_row_col(uint32_t row, uint32_t col)
{
    return (static_cast<uint64_t>(row) << 32) | col;
}

CUDA_CALLABLE uint32_t bsr_get_row(const BsrRowCol& row_col) { return row_col >> 32; }

CUDA_CALLABLE uint32_t bsr_get_col(const BsrRowCol& row_col) { return row_col & INT_MAX; }

template <typename T> struct BsrBlockIsNotZero
{
    int block_size;
    const T* values;

    CUDA_CALLABLE_DEVICE bool operator()(int i) const
    {
        if (!values)
            return true;

        const T* val = values + i * block_size;
        for (int i = 0; i < block_size; ++i, ++val)
        {
            if (*val != T(0))
                return true;
        }
        return false;
    }
};

template <typename T>
__global__ void bsr_fill_triplet_key_values(const int nnz, const int nrow, const int* tpl_rows, const int* tpl_columns,
                                            const BsrBlockIsNotZero<T> nonZero, uint32_t* block_indices,
                                            BsrRowCol* tpl_row_col)
{
    int block = blockIdx.x * blockDim.x + threadIdx.x;
    if (block >= nnz)
        return;

    const int row = tpl_rows[block];
    const int col = tpl_columns[block];
    const bool is_valid = row >= 0 && row < nrow;

    const BsrRowCol row_col = is_valid && nonZero(block) ? bsr_combine_row_col(row, col) : PRUNED_ROWCOL;
    tpl_row_col[block] = row_col;
    block_indices[block] = block;
}

template <typename T>
__global__ void bsr_find_row_offsets(uint32_t row_count, const T* d_nnz, const BsrRowCol* unique_row_col,
                                     int* row_offsets)
{
    const uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row > row_count)
        return;

    const uint32_t nnz = *d_nnz;
    if (row == 0 || nnz == 0)
    {
        row_offsets[row] = 0;
        return;
    }

    if (bsr_get_row(unique_row_col[nnz - 1]) < row)
    {
        row_offsets[row] = nnz;
        return;
    }

    // binary search for row start
    uint32_t lower = 0;
    uint32_t upper = nnz - 1;
    while (lower < upper)
    {
        uint32_t mid = lower + (upper - lower) / 2;

        if (bsr_get_row(unique_row_col[mid]) < row)
        {
            lower = mid + 1;
        }
        else
        {
            upper = mid;
        }
    }

    row_offsets[row] = lower;
}

template <typename T>
__global__ void bsr_merge_blocks(const uint32_t* d_nnz, int block_size, const uint32_t* block_offsets,
                                 const uint32_t* sorted_block_indices, const BsrRowCol* unique_row_cols,
                                 const T* tpl_values, int* bsr_cols, T* bsr_values)

{
    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= *d_nnz)
        return;

    const BsrRowCol row_col = unique_row_cols[i];
    bsr_cols[i] = bsr_get_col(row_col);

    // Accumulate merged block values
    if (row_col == PRUNED_ROWCOL || bsr_values == nullptr)
        return;

    const uint32_t beg = i ? block_offsets[i - 1] : 0;
    const uint32_t end = block_offsets[i];

    T* bsr_val = bsr_values + i * block_size;
    const T* tpl_val = tpl_values + sorted_block_indices[beg] * block_size;

    for (int k = 0; k < block_size; ++k)
    {
        bsr_val[k] = tpl_val[k];
    }

    for (uint32_t cur = beg + 1; cur != end; ++cur)
    {
        const T* tpl_val = tpl_values + sorted_block_indices[cur] * block_size;
        for (int k = 0; k < block_size; ++k)
        {
            bsr_val[k] += tpl_val[k];
        }
    }
}

template <typename T>
void bsr_matrix_from_triplets_device(const int rows_per_block, const int cols_per_block, const int row_count,
                                     const int nnz, const int* tpl_rows, const int* tpl_columns, const T* tpl_values,
                                     const bool prune_numerical_zeros, int* bsr_offsets, int* bsr_columns,
                                     T* bsr_values, int* bsr_nnz, void* bsr_nnz_event)
{
    const int block_size = rows_per_block * cols_per_block;

    void* context = cuda_context_get_current();
    ContextGuard guard(context);

    // Per-context cached temporary buffers
    // BsrFromTripletsTemp& bsr_temp = g_bsr_from_triplets_temp_map[context];

    hipStream_t stream = static_cast<hipStream_t>(cuda_stream_get_current());

    ScopedTemporary<uint32_t> block_indices(context, 2 * nnz + 1);
    ScopedTemporary<BsrRowCol> combined_row_col(context, 2 * nnz);

    hipcub::DoubleBuffer<uint32_t> d_keys(block_indices.buffer(), block_indices.buffer() + nnz);
    hipcub::DoubleBuffer<BsrRowCol> d_values(combined_row_col.buffer(), combined_row_col.buffer() + nnz);

    uint32_t* unique_triplet_count = block_indices.buffer() + 2 * nnz;

    // Combine rows and columns so we can sort on them both
    BsrBlockIsNotZero<T> isNotZero{block_size, prune_numerical_zeros ? tpl_values : nullptr};
    wp_launch_device(WP_CURRENT_CONTEXT, bsr_fill_triplet_key_values, nnz,
                     (nnz, row_count, tpl_rows, tpl_columns, isNotZero, d_keys.Current(), d_values.Current()));

    // Sort
    {
        size_t buff_size = 0;
        check_cuda(hipcub::DeviceRadixSort::SortPairs(nullptr, buff_size, d_values, d_keys, nnz, 0, 64, stream));
        ScopedTemporary<> temp(context, buff_size);
        check_cuda(hipcub::DeviceRadixSort::SortPairs(temp.buffer(), buff_size, d_values, d_keys, nnz, 0, 64, stream));
    }

    // Runlength encode row-col sequences
    {
        size_t buff_size = 0;
        check_cuda(hipcub::DeviceRunLengthEncode::Encode(nullptr, buff_size, d_values.Current(), d_values.Alternate(),
                                                      d_keys.Alternate(), unique_triplet_count, nnz, stream));
        ScopedTemporary<> temp(context, buff_size);
        check_cuda(hipcub::DeviceRunLengthEncode::Encode(temp.buffer(), buff_size, d_values.Current(),
                                                      d_values.Alternate(), d_keys.Alternate(), unique_triplet_count,
                                                      nnz, stream));
    }

    // Compute row offsets from sorted unique blocks
    wp_launch_device(WP_CURRENT_CONTEXT, bsr_find_row_offsets, row_count + 1,
                     (row_count, unique_triplet_count, d_values.Alternate(), bsr_offsets));

    if (bsr_nnz)
    {
        // Copy nnz to host, and record an event for the competed transfer if desired

        memcpy_d2h(WP_CURRENT_CONTEXT, bsr_nnz, bsr_offsets + row_count, sizeof(int), stream);

        if (bsr_nnz_event)
        {
            cuda_event_record(bsr_nnz_event, stream);
        }
    }

    // Scan repeated block counts
    {
        size_t buff_size = 0;
        check_cuda(
            hipcub::DeviceScan::InclusiveSum(nullptr, buff_size, d_keys.Alternate(), d_keys.Alternate(), nnz, stream));
        ScopedTemporary<> temp(context, buff_size);
        check_cuda(hipcub::DeviceScan::InclusiveSum(temp.buffer(), buff_size, d_keys.Alternate(), d_keys.Alternate(), nnz,
                                                 stream));
    }

    // Accumulate repeated blocks and set column indices
    wp_launch_device(WP_CURRENT_CONTEXT, bsr_merge_blocks, nnz,
                     (unique_triplet_count, block_size, d_keys.Alternate(), d_keys.Current(), d_values.Alternate(),
                      tpl_values, bsr_columns, bsr_values));
}

__global__ void bsr_transpose_fill_row_col(const int nnz_upper_bound, const int row_count, const int* bsr_offsets,
                                           const int* bsr_columns, int* block_indices, BsrRowCol* transposed_row_col)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= nnz_upper_bound)
    {
        // Outside of allocated bounds, do nothing
        return;
    }

    if (i >= bsr_offsets[row_count])
    {
        // Below upper bound but above actual nnz count, mark as invalid
        transposed_row_col[i] = PRUNED_ROWCOL;
        return;
    }

    block_indices[i] = i;

    // Binary search for row
    int lower = 0;
    int upper = row_count - 1;

    while (lower < upper)
    {
        int mid = lower + (upper - lower) / 2;

        if (bsr_offsets[mid + 1] <= i)
        {
            lower = mid + 1;
        }
        else
        {
            upper = mid;
        }
    }

    const int row = lower;
    const int col = bsr_columns[i];
    BsrRowCol row_col = bsr_combine_row_col(col, row);
    transposed_row_col[i] = row_col;
}

template <int Rows, int Cols, typename T> struct BsrBlockTransposer
{
    void CUDA_CALLABLE_DEVICE operator()(const T* src, T* dest) const
    {
        for (int r = 0; r < Rows; ++r)
        {
            for (int c = 0; c < Cols; ++c)
            {
                dest[c * Rows + r] = src[r * Cols + c];
            }
        }
    }
};

template <typename T> struct BsrBlockTransposer<-1, -1, T>
{

    int row_count;
    int col_count;

    void CUDA_CALLABLE_DEVICE operator()(const T* src, T* dest) const
    {
        for (int r = 0; r < row_count; ++r)
        {
            for (int c = 0; c < col_count; ++c)
            {
                dest[c * row_count + r] = src[r * col_count + c];
            }
        }
    }
};

template <int Rows, int Cols, typename T>
__global__ void bsr_transpose_blocks(const int* nnz, const int block_size, BsrBlockTransposer<Rows, Cols, T> transposer,
                                     const int* block_indices, const BsrRowCol* transposed_indices, const T* bsr_values,
                                     int* transposed_bsr_columns, T* transposed_bsr_values)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *nnz)
        return;

    const int src_idx = block_indices[i];

    transposer(bsr_values + src_idx * block_size, transposed_bsr_values + i * block_size);

    transposed_bsr_columns[i] = bsr_get_col(transposed_indices[i]);
}

template <typename T>
void launch_bsr_transpose_blocks(int nnz, const int* d_nnz, const int block_size, const int rows_per_block,
                                 const int cols_per_block, const int* block_indices,
                                 const BsrRowCol* transposed_indices, const T* bsr_values, int* transposed_bsr_columns,
                                 T* transposed_bsr_values)
{

    switch (rows_per_block)
    {
    case 1:
        switch (cols_per_block)
        {
        case 1:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<1, 1, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        case 2:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<1, 2, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        case 3:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<1, 3, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        }
    case 2:
        switch (cols_per_block)
        {
        case 1:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<2, 1, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        case 2:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<2, 2, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        case 3:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<2, 3, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        }
    case 3:
        switch (cols_per_block)
        {
        case 1:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<3, 1, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        case 2:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<3, 2, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        case 3:
            wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                             (d_nnz, block_size, BsrBlockTransposer<3, 3, T>{}, block_indices, transposed_indices,
                              bsr_values, transposed_bsr_columns, transposed_bsr_values));
            return;
        }
    }

    wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_blocks, nnz,
                     (d_nnz, block_size, BsrBlockTransposer<-1, -1, T>{rows_per_block, cols_per_block}, block_indices,
                      transposed_indices, bsr_values, transposed_bsr_columns, transposed_bsr_values));
}

template <typename T>
void bsr_transpose_device(int rows_per_block, int cols_per_block, int row_count, int col_count, int nnz,
                          const int* bsr_offsets, const int* bsr_columns, const T* bsr_values,
                          int* transposed_bsr_offsets, int* transposed_bsr_columns, T* transposed_bsr_values)
{

    const int block_size = rows_per_block * cols_per_block;

    void* context = cuda_context_get_current();
    ContextGuard guard(context);

    hipStream_t stream = static_cast<hipStream_t>(cuda_stream_get_current());

    ScopedTemporary<int> block_indices(context, 2 * nnz);
    ScopedTemporary<BsrRowCol> combined_row_col(context, 2 * nnz);

    hipcub::DoubleBuffer<int> d_keys(block_indices.buffer(), block_indices.buffer() + nnz);
    hipcub::DoubleBuffer<BsrRowCol> d_values(combined_row_col.buffer(), combined_row_col.buffer() + nnz);

    wp_launch_device(WP_CURRENT_CONTEXT, bsr_transpose_fill_row_col, nnz,
                     (nnz, row_count, bsr_offsets, bsr_columns, d_keys.Current(), d_values.Current()));

    // Sort blocks
    {
        size_t buff_size = 0;
        check_cuda(hipcub::DeviceRadixSort::SortPairs(nullptr, buff_size, d_values, d_keys, nnz, 0, 64, stream));
        ScopedTemporary<> temp(context, buff_size);
        check_cuda(hipcub::DeviceRadixSort::SortPairs(temp.buffer(), buff_size, d_values, d_keys, nnz, 0, 64, stream));
    }

    // Compute row offsets from sorted unique blocks
    wp_launch_device(WP_CURRENT_CONTEXT, bsr_find_row_offsets, col_count + 1,
                     (col_count, bsr_offsets + row_count, d_values.Current(), transposed_bsr_offsets));

    // Move and transpose individual blocks
    if (transposed_bsr_values != nullptr)
    {
        launch_bsr_transpose_blocks(nnz, bsr_offsets + row_count, block_size, rows_per_block, cols_per_block,
                                    d_keys.Current(), d_values.Current(), bsr_values, transposed_bsr_columns,
                                    transposed_bsr_values);
    }
}

} // namespace

void bsr_matrix_from_triplets_float_device(int rows_per_block, int cols_per_block, int row_count, int nnz,
                                           int* tpl_rows, int* tpl_columns, void* tpl_values,
                                           bool prune_numerical_zeros, int* bsr_offsets, int* bsr_columns,
                                           void* bsr_values, int* bsr_nnz, void* bsr_nnz_event)
{
    return bsr_matrix_from_triplets_device<float>(
        rows_per_block, cols_per_block, row_count, nnz, tpl_rows, tpl_columns, static_cast<const float*>(tpl_values),
        prune_numerical_zeros, bsr_offsets, bsr_columns, static_cast<float*>(bsr_values), bsr_nnz, bsr_nnz_event);
}

void bsr_matrix_from_triplets_double_device(int rows_per_block, int cols_per_block, int row_count, int nnz,
                                            int* tpl_rows, int* tpl_columns, void* tpl_values,
                                            bool prune_numerical_zeros, int* bsr_offsets, int* bsr_columns,
                                            void* bsr_values, int* bsr_nnz, void* bsr_nnz_event)
{
    return bsr_matrix_from_triplets_device<double>(
        rows_per_block, cols_per_block, row_count, nnz, tpl_rows, tpl_columns, static_cast<const double*>(tpl_values),
        prune_numerical_zeros, bsr_offsets, bsr_columns, static_cast<double*>(bsr_values), bsr_nnz, bsr_nnz_event);
}

void bsr_transpose_float_device(int rows_per_block, int cols_per_block, int row_count, int col_count, int nnz,
                                int* bsr_offsets, int* bsr_columns, void* bsr_values, int* transposed_bsr_offsets,
                                int* transposed_bsr_columns, void* transposed_bsr_values)
{
    bsr_transpose_device(rows_per_block, cols_per_block, row_count, col_count, nnz, bsr_offsets, bsr_columns,
                         static_cast<const float*>(bsr_values), transposed_bsr_offsets, transposed_bsr_columns,
                         static_cast<float*>(transposed_bsr_values));
}

void bsr_transpose_double_device(int rows_per_block, int cols_per_block, int row_count, int col_count, int nnz,
                                 int* bsr_offsets, int* bsr_columns, void* bsr_values, int* transposed_bsr_offsets,
                                 int* transposed_bsr_columns, void* transposed_bsr_values)
{
    bsr_transpose_device(rows_per_block, cols_per_block, row_count, col_count, nnz, bsr_offsets, bsr_columns,
                         static_cast<const double*>(bsr_values), transposed_bsr_offsets, transposed_bsr_columns,
                         static_cast<double*>(transposed_bsr_values));
}
