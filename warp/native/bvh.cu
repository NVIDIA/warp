#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "warp.h"
#include "cuda_util.h"
#include "bvh.h"
#include "sort.h"

#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define THRUST_IGNORE_CUB_VERSION_CHECK
#define REORDER_HOST_TREE

#include <hipcub/hipcub.hpp>


namespace wp
{
    void bvh_create_host(vec3* lowers, vec3* uppers, int num_items, int constructor_type, BVH& bvh);
    void bvh_destroy_host(BVH& bvh);

// for LBVH: this will start with some muted leaf nodes, but that is okay, we can still trace up because there parents information is still valid
// the only thing worth mentioning is that when the parent leaf node is also a leaf node, we need to recompute its bounds, since their child information are lost
// for a compact tree such as those from SAH or Median constructor, there is no muted leaf nodes
__global__ void bvh_refit_kernel(int n, const int* __restrict__ parents, int* __restrict__ child_count, int* __restrict__ primitive_indices, BVHPackedNodeHalf* __restrict__ node_lowers, BVHPackedNodeHalf* __restrict__ node_uppers, const vec3* item_lowers, const vec3* item_uppers)
{
    int index = blockDim.x*blockIdx.x + threadIdx.x;

    if (index < n)
    {
        bool leaf = node_lowers[index].b;
        int parent = parents[index];

        if (leaf)
        {
            BVHPackedNodeHalf& lower = node_lowers[index];
            BVHPackedNodeHalf& upper = node_uppers[index];
            // update the leaf node

            // only need to compute bound when this is a valid leaf node
            if (!node_lowers[parent].b)
            {
                const int start = lower.i;
                const int end = upper.i;

                bounds3 bound;
                for (int primitive_counter = start; primitive_counter < end; primitive_counter++)
                {
                    const int primitive = primitive_indices[primitive_counter];
                    bound.add_bounds(item_lowers[primitive], item_uppers[primitive]);
                }
                (vec3&)lower = bound.lower;
                (vec3&)upper = bound.upper;
            }
        }
        else
        {
            // only keep leaf threads
            return;
        }

        // update hierarchy
        for (;;)
        {
            parent = parents[index];
            // reached root
            if (parent == -1)
                return;

            // ensure all writes are visible
            __threadfence();
         
            int finished = atomicAdd(&child_count[parent], 1);

            // if we have are the last thread (such that the parent node is now complete)
            // then update its bounds and move onto the next parent in the hierarchy
            if (finished == 1)
            {
                BVHPackedNodeHalf& parent_lower = node_lowers[parent];
                BVHPackedNodeHalf& parent_upper = node_uppers[parent];
                if (parent_lower.b)
                    // a packed leaf node can still be a parent in LBVH, we need to recompute its bounds
                    // since we've lost its left and right child node index in the muting process
                {
                    // update the leaf node
                    int parent_parent = parents[parent];;

                    // only need to compute bound when this is a valid leaf node
                    if (!node_lowers[parent_parent].b)
                    {
                        const int start = parent_lower.i;
                        const int end = parent_upper.i;
                        bounds3 bound;
                        for (int primitive_counter = start; primitive_counter < end; primitive_counter++)
                        {
                            const int primitive = primitive_indices[primitive_counter];
                            bound.add_bounds(item_lowers[primitive], item_uppers[primitive]);
                        }

                        (vec3&)parent_lower = bound.lower;
                        (vec3&)parent_upper = bound.upper;
                    }
                }
                else
                {
                    const int left_child = parent_lower.i;
                    const int right_child = parent_upper.i;

                    vec3 left_lower = (vec3&)(node_lowers[left_child]);
                    vec3 left_upper = (vec3&)(node_uppers[left_child]);
                    vec3 right_lower = (vec3&)(node_lowers[right_child]);
                    vec3 right_upper = (vec3&)(node_uppers[right_child]);

                    // union of child bounds
                    vec3 lower = min(left_lower, right_lower);
                    vec3 upper = max(left_upper, right_upper);

                    // write new BVH nodes
                    (vec3&)parent_lower = lower;
                    (vec3&)parent_upper = upper;
                }
                // move onto processing the parent
                index = parent;
            }
            else
            {
                // parent not ready (we are the first child), terminate thread
                break;
            }
        }		
    }
}


void bvh_refit_device(BVH& bvh)
{
    ContextGuard guard(bvh.context);

    // clear child counters
    memset_device(WP_CURRENT_CONTEXT, bvh.node_counts, 0, sizeof(int) * bvh.max_nodes);
    wp_launch_device(WP_CURRENT_CONTEXT, bvh_refit_kernel, bvh.num_leaf_nodes, (bvh.num_leaf_nodes, bvh.node_parents, bvh.node_counts, bvh.primitive_indices, bvh.node_lowers, bvh.node_uppers, bvh.item_lowers, bvh.item_uppers));
}


/////////////////////////////////////////////////////////////////////////////////////////////

// Create a linear BVH as described in Fast and Simple Agglomerative LBVH construction
// this is a bottom-up clustering method that outputs one node per-leaf 
//
class LinearBVHBuilderGPU
{
public:

    LinearBVHBuilderGPU();
    ~LinearBVHBuilderGPU();

    // takes a bvh (host ref), and pointers to the GPU lower and upper bounds for each triangle
    void build(BVH& bvh, const vec3* item_lowers, const vec3* item_uppers, int num_items, bounds3* total_bounds);

private:

    // temporary data used during building
    int* indices;
    int* keys;
    int* deltas;
    int* range_lefts;
    int* range_rights;
    int* num_children;

    // bounds data when total item bounds built on GPU
    vec3* total_lower;
    vec3* total_upper;
    vec3* total_inv_edges;
};

////////////////////////////////////////////////////////



__global__ void compute_morton_codes(const vec3* __restrict__ item_lowers, const vec3* __restrict__ item_uppers, int n, const vec3* grid_lower, const vec3* grid_inv_edges, int* __restrict__ indices, int* __restrict__ keys)
{
    const int index = blockDim.x*blockIdx.x + threadIdx.x;

    if (index < n)
    {
        vec3 lower = item_lowers[index];
        vec3 upper = item_uppers[index];

        vec3 center = 0.5f*(lower+upper);

        vec3 local = cw_mul((center-grid_lower[0]), grid_inv_edges[0]);
        
        // 10-bit Morton codes stored in lower 30bits (1024^3 effective resolution)
        int key = morton3<1024>(local[0], local[1], local[2]);

        indices[index] = index;
        keys[index] = key;
    }
}

// calculate the index of the first differing bit between two adjacent Morton keys
__global__ void compute_key_deltas(const int* __restrict__ keys, int* __restrict__ deltas, int n)
{
    const int index = blockDim.x*blockIdx.x + threadIdx.x;

    if (index < n)
    {
        int a = keys[index];
        int b = keys[index+1];

        int x = a^b;
        
        deltas[index] = x;// __clz(x);
    }
}

__global__ void build_leaves(const vec3* __restrict__ item_lowers, const vec3* __restrict__ item_uppers, int n, const int* __restrict__ indices, int* __restrict__ range_lefts, int* __restrict__ range_rights, BVHPackedNodeHalf* __restrict__ lowers, BVHPackedNodeHalf* __restrict__ uppers)
{
    const int index = blockDim.x*blockIdx.x + threadIdx.x;

    if (index < n)
    {
        const int item = indices[index];

        vec3 lower = item_lowers[item];
        vec3 upper = item_uppers[item];

        // write leaf nodes 
        lowers[index] = make_node(lower, item, true);
        uppers[index] = make_node(upper, item, false);

        // write leaf key ranges
        range_lefts[index] = index;
        range_rights[index] = index;
    }
}

// this bottom-up process assigns left and right children and combines bounds to form internal nodes
// there is one thread launched per-leaf node, each thread calculates it's parent node and assigns
// itself to either the left or right parent slot, the last child to complete the parent and moves
// up the hierarchy
__global__ void build_hierarchy(int n, int* root, const int* __restrict__ deltas,  int* __restrict__ num_children, volatile int* __restrict__ range_lefts, volatile int* __restrict__ range_rights, volatile int* __restrict__ parents, volatile BVHPackedNodeHalf* __restrict__ lowers, volatile BVHPackedNodeHalf* __restrict__ uppers)
{
    int index = blockDim.x*blockIdx.x + threadIdx.x;

    if (index < n)
    {
        const int internal_offset = n;

        for (;;)
        {
            int left = range_lefts[index];
            int right = range_rights[index];

            // check if we are the root node, if so then store out our index and terminate
            if (left == 0 && right == n-1)
            {					
                *root = index;
                parents[index] = -1;

                break;
            }

            int childCount = 0;

            int parent;

            if (left == 0 || (right != n-1 && deltas[right] < deltas[left-1]))
            {
                parent = right + internal_offset;

                // set parent left child
                parents[index] = parent;
                lowers[parent].i = index;				
                range_lefts[parent] = left;

                // ensure above writes are visible to all threads
                __threadfence();
                
                childCount = atomicAdd(&num_children[parent], 1);
            }
            else
            {
                parent = left + internal_offset - 1;
                
                // set parent right child
                parents[index] = parent;
                uppers[parent].i = index;
                range_rights[parent] = right;

                // ensure above writes are visible to all threads
                __threadfence();
                
                childCount = atomicAdd(&num_children[parent], 1);
            }

            // if we have are the last thread (such that the parent node is now complete)
            // then update its bounds and move onto the next parent in the hierarchy
            if (childCount == 1)
            {
                const int left_child = lowers[parent].i;
                const int right_child = uppers[parent].i;

                vec3 left_lower = vec3(lowers[left_child].x,
                                       lowers[left_child].y, 
                                       lowers[left_child].z);

                vec3 left_upper = vec3(uppers[left_child].x,
                                       uppers[left_child].y, 
                                       uppers[left_child].z);

                vec3 right_lower = vec3(lowers[right_child].x,
                                        lowers[right_child].y,
                                        lowers[right_child].z);


                vec3 right_upper = vec3(uppers[right_child].x, 
                                        uppers[right_child].y, 
                                        uppers[right_child].z);

                // bounds_union of child bounds
                vec3 lower = min(left_lower, right_lower);
                vec3 upper = max(left_upper, right_upper);
                
                // write new BVH nodes
                make_node(lowers+parent, lower, left_child, false);
                make_node(uppers+parent, upper, right_child, false);

                // move onto processing the parent
                index = parent;
            }
            else
            {
                // parent not ready (we are the first child), terminate thread
                break;
            }
        }		
    }
}

/*
* LBVH uses a bottom-up constructor which makes variable-sized leaf nodes more challenging to achieve. 
* Simply splitting the ordered primitives into uniform groups of size BVH_LEAF_SIZE will result in poor
* quality. Instead, after the hierarchy is built, we convert any intermediate node whose size is 
* <= BVH_LEAF_SIZE into a new leaf node. This process is done using the new kernel function called 
* mark_packed_leaf_nodes .
*/
__global__ void mark_packed_leaf_nodes(int n, volatile int* __restrict__ range_lefts, volatile int* __restrict__ range_rights,
    volatile BVHPackedNodeHalf* __restrict__ lowers, volatile BVHPackedNodeHalf* __restrict__ uppers)
{
    int node_index = blockDim.x * blockIdx.x + threadIdx.x;
    if (node_index < n)
    {
        // mark the node as leaf if its range is less than LEAF_SIZE_LBVH
        // this will forever mute its child nodes so that they will never be accessed

        int left = range_lefts[node_index];
        // the LBVH constructor's range is defined as left <= i <= right
        // we need to convert it to our convention: left <= i < right
        int right = range_rights[node_index] + 1;
        // printf("node %d (left %d right %d)", node_index, left, right);
        if (right - left <= BVH_LEAF_SIZE)
        {
            lowers[node_index].b = 1;
            lowers[node_index].i = left;
            uppers[node_index].i = right;

            // printf("node %d (left %d right %d) is set to child\n", node_index, left, right);
        }
    }
}


CUDA_CALLABLE inline vec3 Vec3Max(const vec3& a, const vec3& b) { return wp::max(a, b); }
CUDA_CALLABLE inline vec3 Vec3Min(const vec3& a, const vec3& b) { return wp::min(a, b); }

__global__ void compute_total_bounds(const vec3* item_lowers, const vec3* item_uppers, vec3* total_lower, vec3* total_upper, int num_items)
{
     typedef hipcub::BlockReduce<vec3, 256> BlockReduce;

     __shared__ typename BlockReduce::TempStorage temp_storage;

     const int blockStart = blockDim.x*blockIdx.x;
     const int numValid = ::min(num_items-blockStart, blockDim.x);

     const int tid = blockStart + threadIdx.x;

     if (tid < num_items)
     {
        vec3 lower = item_lowers[tid];
        vec3 upper = item_uppers[tid];

         vec3 block_upper = BlockReduce(temp_storage).Reduce(upper, Vec3Max, numValid);

         // sync threads because second reduce uses same temp storage as first
         __syncthreads();

         vec3 block_lower = BlockReduce(temp_storage).Reduce(lower, Vec3Min, numValid);

         if (threadIdx.x == 0)
         {
             // write out block results, expanded by the radius
             atomic_max(total_upper, block_upper);
             atomic_min(total_lower, block_lower);
         }	 
    }
}

// compute inverse edge length, this is just done on the GPU to avoid a CPU->GPU sync point
__global__ void compute_total_inv_edges(const vec3* total_lower, const vec3* total_upper, vec3* total_inv_edges)
{
    vec3 edges = (total_upper[0]-total_lower[0]);
    edges += vec3(0.0001f);

    total_inv_edges[0] = vec3(1.0f/edges[0], 1.0f/edges[1], 1.0f/edges[2]);
}



LinearBVHBuilderGPU::LinearBVHBuilderGPU() 
    : indices(NULL)
    , keys(NULL)
    , deltas(NULL)
    , range_lefts(NULL)
    , range_rights(NULL)
    , num_children(NULL)
    , total_lower(NULL)
    , total_upper(NULL)
    , total_inv_edges(NULL)
{
    total_lower = (vec3*)alloc_device(WP_CURRENT_CONTEXT, sizeof(vec3));
    total_upper = (vec3*)alloc_device(WP_CURRENT_CONTEXT, sizeof(vec3));
    total_inv_edges = (vec3*)alloc_device(WP_CURRENT_CONTEXT, sizeof(vec3));
}

LinearBVHBuilderGPU::~LinearBVHBuilderGPU()
{
    free_device(WP_CURRENT_CONTEXT, total_lower);
    free_device(WP_CURRENT_CONTEXT, total_upper);
    free_device(WP_CURRENT_CONTEXT, total_inv_edges);
}



void LinearBVHBuilderGPU::build(BVH& bvh, const vec3* item_lowers, const vec3* item_uppers, int num_items, bounds3* total_bounds)
{
    // allocate temporary memory used during  building
    indices = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int)*num_items*2); 	// *2 for radix sort
    keys = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int)*num_items*2);	    // *2 for radix sort
    deltas = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int)*num_items);    	// highest differentiating bit between keys for item i and i+1
    range_lefts = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int)*bvh.max_nodes);
    range_rights = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int)*bvh.max_nodes);
    num_children = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int)*bvh.max_nodes);

    // if total bounds supplied by the host then we just 
    // compute our edge length and upload it to the GPU directly
    if (total_bounds)
    {
        // calculate Morton codes
        vec3 edges = (*total_bounds).edges();
        edges += vec3(0.0001f);

        vec3 inv_edges = vec3(1.0f/edges[0], 1.0f/edges[1], 1.0f/edges[2]);
        
        memcpy_h2d(WP_CURRENT_CONTEXT, total_lower, &total_bounds->lower[0], sizeof(vec3));
        memcpy_h2d(WP_CURRENT_CONTEXT, total_upper, &total_bounds->upper[0], sizeof(vec3));
        memcpy_h2d(WP_CURRENT_CONTEXT, total_inv_edges, &inv_edges[0], sizeof(vec3));
    }
    else
    {
        static vec3 upper(-FLT_MAX);
        static vec3 lower(FLT_MAX);

        memcpy_h2d(WP_CURRENT_CONTEXT, total_lower, &lower, sizeof(lower));
        memcpy_h2d(WP_CURRENT_CONTEXT, total_upper, &upper, sizeof(upper));

        // compute the total bounds on the GPU
        wp_launch_device(WP_CURRENT_CONTEXT, compute_total_bounds, num_items, (item_lowers, item_uppers, total_lower, total_upper, num_items));

        // compute the total edge length
        wp_launch_device(WP_CURRENT_CONTEXT, compute_total_inv_edges, 1, (total_lower, total_upper, total_inv_edges));
    }

    // assign 30-bit Morton code based on the centroid of each triangle and bounds for each leaf
    wp_launch_device(WP_CURRENT_CONTEXT, compute_morton_codes, num_items, (item_lowers, item_uppers, num_items, total_lower, total_inv_edges, indices, keys));
    
    // sort items based on Morton key (note the 32-bit sort key corresponds to the template parameter to morton3, i.e. 3x9 bit keys combined)
    radix_sort_pairs_device(WP_CURRENT_CONTEXT, keys, indices, num_items);
    memcpy_d2d(WP_CURRENT_CONTEXT, bvh.primitive_indices, indices, sizeof(int) * num_items);

    // calculate deltas between adjacent keys
    wp_launch_device(WP_CURRENT_CONTEXT, compute_key_deltas, num_items, (keys, deltas, num_items-1));

    // initialize leaf nodes
    wp_launch_device(WP_CURRENT_CONTEXT, build_leaves, num_items, (item_lowers, item_uppers, num_items, indices, range_lefts, range_rights, bvh.node_lowers, bvh.node_uppers));
    
    // reset children count, this is our atomic counter so we know when an internal node is complete, only used during building
    memset_device(WP_CURRENT_CONTEXT, num_children, 0, sizeof(int)*bvh.max_nodes);

    // build the tree and internal node bounds
    wp_launch_device(WP_CURRENT_CONTEXT, build_hierarchy, num_items, (num_items, bvh.root, deltas, num_children, range_lefts, range_rights, bvh.node_parents, bvh.node_lowers, bvh.node_uppers));
    wp_launch_device(WP_CURRENT_CONTEXT, mark_packed_leaf_nodes, bvh.max_nodes, (bvh.max_nodes, range_lefts, range_rights, bvh.node_lowers, bvh.node_uppers));

    // free temporary memory
    free_device(WP_CURRENT_CONTEXT, indices);
    free_device(WP_CURRENT_CONTEXT, keys);
    free_device(WP_CURRENT_CONTEXT, deltas);

    free_device(WP_CURRENT_CONTEXT, range_lefts);
    free_device(WP_CURRENT_CONTEXT, range_rights);
    free_device(WP_CURRENT_CONTEXT, num_children);

}

// buffer_size is the number of T, not the number of bytes
template<typename T>
T* make_device_buffer_of(void* context, T* host_buffer, size_t buffer_size)
{
    T* device_buffer = (T*)alloc_device(context, sizeof(T) * buffer_size);;
    memcpy_h2d(context, device_buffer, host_buffer, sizeof(T) * buffer_size);

    return device_buffer;
}

void copy_host_tree_to_device(void* context, BVH& bvh_host, BVH& bvh_device_on_host)
{
#ifdef REORDER_HOST_TREE


    // reorder bvh_host such that its nodes are in the front
    // this is essential for the device refit 
    BVHPackedNodeHalf* node_lowers_reordered = new BVHPackedNodeHalf[bvh_host.max_nodes];
    BVHPackedNodeHalf* node_uppers_reordered = new BVHPackedNodeHalf[bvh_host.max_nodes];

    int* node_parents_reordered = new int[bvh_host.max_nodes];

    std::vector<int> old_to_new(bvh_host.max_nodes, -1);

    // We will place nodes in this order:
    //   Pass 1: leaf nodes (except if it's the root index)
    //   Pass 2: non-leaf, non-root
    //   Pass 3: root node
    int next_pos = 0;

    const int root_index = *bvh_host.root;
    // Pass 1: place leaf nodes at the front 
    for (int i = 0; i < bvh_host.num_nodes; ++i)
    {
        if (bvh_host.node_lowers[i].b)
        {
            node_lowers_reordered[next_pos] = bvh_host.node_lowers[i];
            node_uppers_reordered[next_pos] = bvh_host.node_uppers[i];
            old_to_new[i] = next_pos;
            next_pos++;
        }
    }

    // Pass 2: place non-leaf, non-root nodes
    for (int i = 0; i < bvh_host.num_nodes; ++i)
    {
        if (i == root_index)
        {
            if (bvh_host.node_lowers[i].b)
                // if root node is leaf node, there must be only be one node
            {
                *bvh_host.root = 0;
            }
            else
            {
                *bvh_host.root = next_pos;
            }
        }
        if (!bvh_host.node_lowers[i].b)
        {
            node_lowers_reordered[next_pos] = bvh_host.node_lowers[i];
            node_uppers_reordered[next_pos] = bvh_host.node_uppers[i];
            old_to_new[i] = next_pos;
            next_pos++;
        }
    }

    // We can do that by enumerating all old->new pairs:
    for (int old_index = 0; old_index < bvh_host.num_nodes; ++old_index) {
        int new_index = old_to_new[old_index];  // new index

        int old_parent = bvh_host.node_parents[old_index];
        if (old_parent != -1)
        {
            node_parents_reordered[new_index] = old_to_new[old_parent];
        }
        else
        {
            node_parents_reordered[new_index] = -1;
        }

        // only need to modify the child index of non-leaf nodes
        if (!bvh_host.node_lowers[old_index].b)
        {
            node_lowers_reordered[new_index].i = old_to_new[bvh_host.node_lowers[old_index].i];
            node_uppers_reordered[new_index].i = old_to_new[bvh_host.node_uppers[old_index].i];
        }
    }

    delete[] bvh_host.node_lowers;
    delete[] bvh_host.node_uppers;
    delete[] bvh_host.node_parents;

    bvh_host.node_lowers = node_lowers_reordered;
    bvh_host.node_uppers = node_uppers_reordered;
    bvh_host.node_parents = node_parents_reordered;
#endif // REORDER_HOST_TREE

    bvh_device_on_host.num_nodes = bvh_host.num_nodes;
    bvh_device_on_host.num_leaf_nodes = bvh_host.num_leaf_nodes;
    bvh_device_on_host.max_nodes = bvh_host.max_nodes;
    bvh_device_on_host.num_items = bvh_host.num_items;
    bvh_device_on_host.max_depth = bvh_host.max_depth;

    bvh_device_on_host.root = (int*)alloc_device(context, sizeof(int));
    memcpy_h2d(context, bvh_device_on_host.root, bvh_host.root, sizeof(int));
    bvh_device_on_host.context = context;

    bvh_device_on_host.node_lowers = make_device_buffer_of(context, bvh_host.node_lowers, bvh_host.max_nodes);
    bvh_device_on_host.node_uppers = make_device_buffer_of(context, bvh_host.node_uppers, bvh_host.max_nodes);
    bvh_device_on_host.node_parents = make_device_buffer_of(context, bvh_host.node_parents, bvh_host.max_nodes);
    bvh_device_on_host.primitive_indices = make_device_buffer_of(context, bvh_host.primitive_indices, bvh_host.num_items);
}

// create in-place given existing descriptor
void bvh_create_device(void* context, vec3* lowers, vec3* uppers, int num_items, int constructor_type, BVH& bvh_device_on_host)
{
    ContextGuard guard(context);
    if (constructor_type == BVH_CONSTRUCTOR_SAH || constructor_type == BVH_CONSTRUCTOR_MEDIAN)
        // CPU based constructors
    {
        // copy bounds back to CPU
        std::vector<vec3> lowers_host(num_items);
        std::vector<vec3> uppers_host(num_items);
        memcpy_d2h(WP_CURRENT_CONTEXT, lowers_host.data(), lowers, sizeof(vec3) * num_items);
        memcpy_d2h(WP_CURRENT_CONTEXT, uppers_host.data(), uppers, sizeof(vec3) * num_items);

        // run CPU based constructor
        wp::BVH bvh_host;
        bvh_create_host(lowers_host.data(), uppers_host.data(), num_items, constructor_type, bvh_host);

        // copy host tree to device
        wp::copy_host_tree_to_device(WP_CURRENT_CONTEXT, bvh_host, bvh_device_on_host);
        // replace host bounds with device bounds
        bvh_device_on_host.item_lowers = lowers;
        bvh_device_on_host.item_uppers = uppers;
        // node_counts is not allocated for host tree
        bvh_device_on_host.node_counts = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int) * bvh_device_on_host.max_nodes);
        bvh_destroy_host(bvh_host);
    }
    else if (constructor_type == BVH_CONSTRUCTOR_LBVH)
    {
        bvh_device_on_host.num_items = num_items;
        bvh_device_on_host.max_nodes = 2 * num_items;
        bvh_device_on_host.num_leaf_nodes = num_items;
        bvh_device_on_host.node_lowers = (BVHPackedNodeHalf*)alloc_device(WP_CURRENT_CONTEXT, sizeof(BVHPackedNodeHalf) * bvh_device_on_host.max_nodes);
        memset_device(WP_CURRENT_CONTEXT, bvh_device_on_host.node_lowers, 0, sizeof(BVHPackedNodeHalf) * bvh_device_on_host.max_nodes);
        bvh_device_on_host.node_uppers = (BVHPackedNodeHalf*)alloc_device(WP_CURRENT_CONTEXT, sizeof(BVHPackedNodeHalf) * bvh_device_on_host.max_nodes);
        memset_device(WP_CURRENT_CONTEXT, bvh_device_on_host.node_uppers, 0, sizeof(BVHPackedNodeHalf) * bvh_device_on_host.max_nodes);
        bvh_device_on_host.node_parents = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int) * bvh_device_on_host.max_nodes);
        bvh_device_on_host.node_counts = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int) * bvh_device_on_host.max_nodes);
        bvh_device_on_host.root = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int));
        bvh_device_on_host.primitive_indices = (int*)alloc_device(WP_CURRENT_CONTEXT, sizeof(int) * num_items);
        bvh_device_on_host.item_lowers = lowers;
        bvh_device_on_host.item_uppers = uppers;

        bvh_device_on_host.context = context ? context : cuda_context_get_current();

        LinearBVHBuilderGPU builder;
        builder.build(bvh_device_on_host, lowers, uppers, num_items, NULL);
    }
    else
    {
        printf("Unrecognized Constructor type: %d! For GPU constructor it should be SAH (0), Median (1), or LBVH (2)!\n", constructor_type);
    }
}

void bvh_destroy_device(BVH& bvh)
{
    ContextGuard guard(bvh.context);

    free_device(WP_CURRENT_CONTEXT, bvh.node_lowers); bvh.node_lowers = NULL;
    free_device(WP_CURRENT_CONTEXT, bvh.node_uppers); bvh.node_uppers = NULL;
    free_device(WP_CURRENT_CONTEXT, bvh.node_parents); bvh.node_parents = NULL;
    free_device(WP_CURRENT_CONTEXT, bvh.node_counts); bvh.node_counts = NULL;
    free_device(WP_CURRENT_CONTEXT, bvh.primitive_indices); bvh.primitive_indices = NULL;
    free_device(WP_CURRENT_CONTEXT, bvh.root); bvh.root = NULL;
}


} // namespace wp


void bvh_refit_device(uint64_t id)
{
    wp::BVH bvh;
    if (bvh_get_descriptor(id, bvh))
    {
        ContextGuard guard(bvh.context);

        bvh_refit_device(bvh);
    }
}

/*
* Since we don't even know the number of true leaf nodes, never mention where they are, we will launch
* the num_items threads, which are identical to the number of leaf nodes in the original tree. The 
* refitting threads will start from the nodes corresponding to the original leaf nodes, which might be 
* muted. However, the muted leaf nodes will still have the pointer to their parents, thus the up-tracing
* can still work. We will only compute the bounding box of a leaf node if its parent is not a leaf node.
*/
uint64_t bvh_create_device(void* context, wp::vec3* lowers, wp::vec3* uppers, int num_items, int constructor_type)
{
    ContextGuard guard(context);
    wp::BVH bvh_device_on_host;
    wp::BVH* bvh_device_ptr = nullptr;
    
    bvh_create_device(WP_CURRENT_CONTEXT, lowers, uppers, num_items, constructor_type, bvh_device_on_host);

    // create device-side BVH descriptor
    bvh_device_ptr = (wp::BVH*)alloc_device(WP_CURRENT_CONTEXT, sizeof(wp::BVH));
    memcpy_h2d(WP_CURRENT_CONTEXT, bvh_device_ptr, &bvh_device_on_host, sizeof(wp::BVH));

    uint64_t bvh_id = (uint64_t)bvh_device_ptr;
    wp::bvh_add_descriptor(bvh_id, bvh_device_on_host);
    return bvh_id;
}


void bvh_destroy_device(uint64_t id)
{
    wp::BVH bvh;
    if (wp::bvh_get_descriptor(id, bvh))
    {
        wp::bvh_destroy_device(bvh);
        wp::bvh_rem_descriptor(id);

        // free descriptor
        free_device(WP_CURRENT_CONTEXT, (void*)id);
    }
}
