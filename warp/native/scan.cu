/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "warp.h"
#include "scan.h"

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <cub/device/device_scan.cuh>

template<typename T>
void scan_device(const T* values_in, T* values_out, int n, bool inclusive)
{
    ContextGuard guard(cuda_context_get_current());

    hipStream_t stream = static_cast<hipStream_t>(cuda_stream_get_current());

    // compute temporary memory required
	size_t scan_temp_size;
    if (inclusive) {
        check_cuda(hipcub::DeviceScan::InclusiveSum(NULL, scan_temp_size, values_in, values_out, n));
    } else {
        check_cuda(hipcub::DeviceScan::ExclusiveSum(NULL, scan_temp_size, values_in, values_out, n));
    }

    void* temp_buffer = alloc_device(WP_CURRENT_CONTEXT, scan_temp_size);

    // scan
    if (inclusive) {
        check_cuda(hipcub::DeviceScan::InclusiveSum(temp_buffer, scan_temp_size, values_in, values_out, n, stream));
    } else {
        check_cuda(hipcub::DeviceScan::ExclusiveSum(temp_buffer, scan_temp_size, values_in, values_out, n, stream));
    }

    free_device(WP_CURRENT_CONTEXT, temp_buffer);
}

template void scan_device(const int*, int*, int, bool);
template void scan_device(const float*, float*, int, bool);
