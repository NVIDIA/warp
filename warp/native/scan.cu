#include "warp.h"
#include "scan.h"

#include "temp_buffer.h"

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <cub/device/device_scan.cuh>

template<typename T>
void scan_device(const T* values_in, T* values_out, int n, bool inclusive)
{
    void *context = cuda_context_get_current();
    TemporaryBuffer &cub_temp = g_temp_buffer_map[context];

    ContextGuard guard(context);

    hipStream_t stream = static_cast<hipStream_t>(cuda_stream_get_current());

    // compute temporary memory required
	size_t scan_temp_size;
    if (inclusive) {
        hipcub::DeviceScan::InclusiveSum(NULL, scan_temp_size, values_in, values_out, n);
    } else {
        hipcub::DeviceScan::ExclusiveSum(NULL, scan_temp_size, values_in, values_out, n);
    }

    cub_temp.ensure_fits(scan_temp_size);

    // scan
    if (inclusive) {
        hipcub::DeviceScan::InclusiveSum(cub_temp.buffer, scan_temp_size, values_in, values_out, n, (hipStream_t)cuda_stream_get_current());
    } else {
        hipcub::DeviceScan::ExclusiveSum(cub_temp.buffer, scan_temp_size, values_in, values_out, n, (hipStream_t)cuda_stream_get_current());
    }
}

template void scan_device(const int*, int*, int, bool);
template void scan_device(const float*, float*, int, bool);
