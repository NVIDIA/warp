/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "warp.h"
#include "cuda_util.h"
#include "sort.h"

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <hipcub/hipcub.hpp>

#include <unordered_map>

// temporary buffer for radix sort
struct RadixSortTemp
{
    void* mem = NULL;
    size_t size = 0;
};

// use unique temp buffers per CUDA stream to avoid race conditions
static std::unordered_map<void*, RadixSortTemp> g_radix_sort_temp_map;


template <typename KeyType>
void radix_sort_reserve_internal(void* context, int n, void** mem_out, size_t* size_out)
{
    ContextGuard guard(context);

    hipcub::DoubleBuffer<KeyType> d_keys;
	hipcub::DoubleBuffer<int> d_values;

    hipStream_t stream = static_cast<hipStream_t>(wp_cuda_stream_get_current());

    // compute temporary memory required
	size_t sort_temp_size;
    check_cuda(hipcub::DeviceRadixSort::SortPairs(
        NULL,
        sort_temp_size,
        d_keys,
        d_values,
        n, 0, sizeof(KeyType)*8,
        stream));

    RadixSortTemp& temp = g_radix_sort_temp_map[stream];

    if (sort_temp_size > temp.size)
    {
	    wp_free_device(WP_CURRENT_CONTEXT, temp.mem);
        temp.mem = wp_alloc_device(WP_CURRENT_CONTEXT, sort_temp_size);
        temp.size = sort_temp_size;
    }
    
    if (mem_out)
        *mem_out = temp.mem;
    if (size_out)
        *size_out = temp.size;
}

void radix_sort_reserve(void* context, int n, void** mem_out, size_t* size_out)
{
    radix_sort_reserve_internal<int>(context, n, mem_out, size_out);
}

void radix_sort_release(void* context, void* stream)
{
    // release temporary buffer for the given stream, if it exists
    auto it = g_radix_sort_temp_map.find(stream);
    if (it != g_radix_sort_temp_map.end())
    {
        wp_free_device(context, it->second.mem);
        g_radix_sort_temp_map.erase(it);
    }
}

template <typename KeyType>
void radix_sort_pairs_device(void* context, KeyType* keys, int* values, int n)
{
    ContextGuard guard(context);

    hipcub::DoubleBuffer<KeyType> d_keys(keys, keys + n);
	hipcub::DoubleBuffer<int> d_values(values, values + n);

    RadixSortTemp temp;
    radix_sort_reserve_internal<KeyType>(WP_CURRENT_CONTEXT, n, &temp.mem, &temp.size);

    // sort
    check_cuda(hipcub::DeviceRadixSort::SortPairs(
        temp.mem,
        temp.size,
        d_keys, 
        d_values, 
        n, 0, sizeof(KeyType)*8, 
        (hipStream_t)wp_cuda_stream_get_current()));

	if (d_keys.Current() != keys)
		wp_memcpy_d2d(WP_CURRENT_CONTEXT, keys, d_keys.Current(), sizeof(KeyType)*n);

	if (d_values.Current() != values)
		wp_memcpy_d2d(WP_CURRENT_CONTEXT, values, d_values.Current(), sizeof(int)*n);
}

void radix_sort_pairs_device(void* context, int* keys, int* values, int n)
{
    radix_sort_pairs_device<int>(context, keys, values, n);
}

void radix_sort_pairs_device(void* context, float* keys, int* values, int n)
{
    radix_sort_pairs_device<float>(context, keys, values, n);
}

void radix_sort_pairs_device(void* context, int64_t* keys, int* values, int n)
{
    radix_sort_pairs_device<int64_t>(context, keys, values, n);
}

void wp_radix_sort_pairs_int_device(uint64_t keys, uint64_t values, int n)
{
    radix_sort_pairs_device(
        WP_CURRENT_CONTEXT,
        reinterpret_cast<int *>(keys),
        reinterpret_cast<int *>(values), n);
}

void wp_radix_sort_pairs_float_device(uint64_t keys, uint64_t values, int n)
{
    radix_sort_pairs_device(
        WP_CURRENT_CONTEXT,
        reinterpret_cast<float *>(keys),
        reinterpret_cast<int *>(values), n);
}

void wp_radix_sort_pairs_int64_device(uint64_t keys, uint64_t values, int n)
{
    radix_sort_pairs_device(
        WP_CURRENT_CONTEXT,
        reinterpret_cast<int64_t *>(keys),
        reinterpret_cast<int *>(values), n);
}

void segmented_sort_reserve(void* context, int n, int num_segments, void** mem_out, size_t* size_out)
{
    ContextGuard guard(context);

    hipcub::DoubleBuffer<int> d_keys;
	hipcub::DoubleBuffer<int> d_values;

    int* start_indices = NULL;
    int* end_indices = NULL;

    hipStream_t stream = static_cast<hipStream_t>(wp_cuda_stream_get_current());

    // compute temporary memory required
	size_t sort_temp_size;
    check_cuda(hipcub::DeviceSegmentedRadixSort::SortPairs(
        NULL,
        sort_temp_size,
        d_keys,
        d_values,
        n, 
        num_segments,
        start_indices,
        end_indices,
        0,
        32,
        stream));

    RadixSortTemp& temp = g_radix_sort_temp_map[stream];

    if (sort_temp_size > temp.size)
    {
	    wp_free_device(WP_CURRENT_CONTEXT, temp.mem);
        temp.mem = wp_alloc_device(WP_CURRENT_CONTEXT, sort_temp_size);
        temp.size = sort_temp_size;
    }
    
    if (mem_out)
        *mem_out = temp.mem;
    if (size_out)
        *size_out = temp.size;
}

// segment_start_indices and segment_end_indices are arrays of length num_segments, where segment_start_indices[i] is the index of the first element 
// in the i-th segment and segment_end_indices[i] is the index after the last element in the i-th segment
// https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceSegmentedRadixSort.html
void segmented_sort_pairs_device(void* context, float* keys, int* values, int n, int* segment_start_indices, int* segment_end_indices, int num_segments)
{
    ContextGuard guard(context);

    hipcub::DoubleBuffer<float> d_keys(keys, keys + n);
	hipcub::DoubleBuffer<int> d_values(values, values + n);

    RadixSortTemp temp;
    segmented_sort_reserve(WP_CURRENT_CONTEXT, n, num_segments, &temp.mem, &temp.size);

    // sort
    check_cuda(hipcub::DeviceSegmentedRadixSort::SortPairs(
        temp.mem,
        temp.size,
        d_keys, 
        d_values, 
        n,
        num_segments,
        segment_start_indices,
        segment_end_indices,
        0,
        32,
        (hipStream_t)wp_cuda_stream_get_current()));

	if (d_keys.Current() != keys)
		wp_memcpy_d2d(WP_CURRENT_CONTEXT, keys, d_keys.Current(), sizeof(float)*n);

	if (d_values.Current() != values)
		wp_memcpy_d2d(WP_CURRENT_CONTEXT, values, d_values.Current(), sizeof(int)*n);
}

void wp_segmented_sort_pairs_float_device(uint64_t keys, uint64_t values, int n, uint64_t segment_start_indices, uint64_t segment_end_indices, int num_segments)
{
    segmented_sort_pairs_device(
        WP_CURRENT_CONTEXT,
        reinterpret_cast<float *>(keys),
        reinterpret_cast<int *>(values), n,
        reinterpret_cast<int *>(segment_start_indices),
        reinterpret_cast<int *>(segment_end_indices),
        num_segments);
}

// segment_indices is an array of length num_segments + 1, where segment_indices[i] is the index of the first element in the i-th segment
// The end of a segment is given by segment_indices[i+1]
// https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceSegmentedSort.html#a-simple-example
void segmented_sort_pairs_device(void* context, int* keys, int* values, int n, int* segment_start_indices, int* segment_end_indices, int num_segments)
{
    ContextGuard guard(context);

    hipcub::DoubleBuffer<int> d_keys(keys, keys + n);
	hipcub::DoubleBuffer<int> d_values(values, values + n);

    RadixSortTemp temp;
    segmented_sort_reserve(WP_CURRENT_CONTEXT, n, num_segments, &temp.mem, &temp.size);

    // sort
    check_cuda(hipcub::DeviceSegmentedRadixSort::SortPairs(
        temp.mem,
        temp.size,
        d_keys, 
        d_values, 
        n,
        num_segments,
        segment_start_indices,
        segment_end_indices,
        0,
        32,
        (hipStream_t)wp_cuda_stream_get_current()));

	if (d_keys.Current() != keys)
		wp_memcpy_d2d(WP_CURRENT_CONTEXT, keys, d_keys.Current(), sizeof(float)*n);

	if (d_values.Current() != values)
		wp_memcpy_d2d(WP_CURRENT_CONTEXT, values, d_values.Current(), sizeof(int)*n);
}

void wp_segmented_sort_pairs_int_device(uint64_t keys, uint64_t values, int n, uint64_t segment_start_indices, uint64_t segment_end_indices, int num_segments)
{
    segmented_sort_pairs_device(
        WP_CURRENT_CONTEXT,
        reinterpret_cast<int *>(keys),
        reinterpret_cast<int *>(values), n,
        reinterpret_cast<int *>(segment_start_indices),
        reinterpret_cast<int *>(segment_end_indices),
        num_segments);
}
