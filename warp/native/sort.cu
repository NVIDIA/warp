/** Copyright (c) 2022 NVIDIA CORPORATION.  All rights reserved.
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

#include "warp.h"
#include "cuda_util.h"
#include "sort.h"

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <hipcub/hipcub.hpp>

#include <map>

// temporary buffer for radix sort
struct RadixSortTemp
{
    void* mem = NULL;
    size_t size = 0;
};

// map temp buffers to CUDA contexts
static std::map<void*, RadixSortTemp> g_radix_sort_temp_map;


void radix_sort_reserve(void* context, int n, void** mem_out, size_t* size_out)
{
    ContextGuard guard(context);

    hipcub::DoubleBuffer<int> d_keys;
	hipcub::DoubleBuffer<int> d_values;

    // compute temporary memory required
	size_t sort_temp_size;
    check_cuda(hipcub::DeviceRadixSort::SortPairs(
        NULL,
        sort_temp_size,
        d_keys,
        d_values,
        n, 0, 32,
        (hipStream_t)cuda_stream_get_current()));

    if (!context)
        context = cuda_context_get_current();

    RadixSortTemp& temp = g_radix_sort_temp_map[context];

    if (sort_temp_size > temp.size)
    {
	    free_device(WP_CURRENT_CONTEXT, temp.mem);
        temp.mem = alloc_device(WP_CURRENT_CONTEXT, sort_temp_size);
        temp.size = sort_temp_size;
    }
    
    if (mem_out)
        *mem_out = temp.mem;
    if (size_out)
        *size_out = temp.size;
}

void radix_sort_pairs_device(void* context, int* keys, int* values, int n)
{
    ContextGuard guard(context);

    hipcub::DoubleBuffer<int> d_keys(keys, keys + n);
	hipcub::DoubleBuffer<int> d_values(values, values + n);

    RadixSortTemp temp;
    radix_sort_reserve(WP_CURRENT_CONTEXT, n, &temp.mem, &temp.size);

    // sort
    check_cuda(hipcub::DeviceRadixSort::SortPairs(
        temp.mem,
        temp.size,
        d_keys, 
        d_values, 
        n, 0, 32, 
        (hipStream_t)cuda_stream_get_current()));

	if (d_keys.Current() != keys)
		memcpy_d2d(WP_CURRENT_CONTEXT, keys, d_keys.Current(), sizeof(int)*n);

	if (d_values.Current() != values)
		memcpy_d2d(WP_CURRENT_CONTEXT, values, d_values.Current(), sizeof(int)*n);
}

void radix_sort_pairs_int_device(uint64_t keys, uint64_t values, int n)
{
    radix_sort_pairs_device(
        WP_CURRENT_CONTEXT,
        reinterpret_cast<int *>(keys),
        reinterpret_cast<int *>(values), n);
}
